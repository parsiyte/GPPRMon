#include "hip/hip_runtime.h"
// Copyright (c) 2019, Xuhao Chen
#include "kcl.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#define USE_SIMPLE
#define USE_BASE_TYPES
#include "gpu_mining/miner.cuh"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
typedef unsigned long long AccType;
typedef uint64_t emb_index_t;

#define USE_SHM
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

__global__ void extend_alloc(size_t begin, size_t end, 
                             unsigned level, unsigned max_size, 
                             GraphGPU graph, EmbeddingList emb_list, 
                             emb_index_t *num_new_emb, AccType *total) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ typename BlockReduce::TempStorage temp_storage;
#ifdef USE_SHM
  __shared__ IndexT emb[BLOCK_SIZE][MAX_SIZE];
#else
  IndexT emb[MAX_SIZE];
#endif
  AccType local_num = 0;
  if(pos < end - begin) {
#ifdef USE_SHM
    emb_list.get_embedding(level, begin + pos, emb[tid]);
#else
    emb_list.get_embedding(level, begin + pos, emb);
#endif
    auto vid = emb_list.get_vid(level, begin + pos);
    auto row_begin = graph.edge_begin(vid);
    auto row_end = graph.edge_end(vid);
    num_new_emb[pos] = 0;
    for (auto e = row_begin; e < row_end; e++) {
      auto dst = graph.getEdgeDst(e);
#ifdef USE_SHM
      if (is_all_connected_dag(dst, emb[tid], level, graph)) {
#else
      if (is_all_connected_dag(dst, emb, level, graph)) {
#endif
        if (level < max_size-2) num_new_emb[pos] ++;
        else local_num += 1;
      }
    }
  }
  AccType block_num = BlockReduce(temp_storage).Sum(local_num);
  if(threadIdx.x == 0) atomicAdd(total, block_num);
}

__global__ void extend_alloc_lb(size_t begin, size_t end, unsigned level, unsigned max_size, 
                                GraphGPU graph, EmbeddingList emb_list, 
                                emb_index_t *num_new_emb, AccType *total) {
  //expandByCta(m, row_offsets, column_indices, depths, in_queue, out_queue, depth);
  //expandByWarp(m, row_offsets, column_indices, depths, in_queue, out_queue, depth);
  unsigned tid = threadIdx.x;
  unsigned base_id = blockIdx.x * blockDim.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ typename BlockReduce::TempStorage reduce_storage;

  const unsigned SCRATCHSIZE = BLOCK_SIZE;
  __shared__ BlockScan::TempStorage temp_storage;
  __shared__ int gather_offsets[SCRATCHSIZE];
  __shared__ unsigned src[SCRATCHSIZE];
  __shared__ IndexT emb[BLOCK_SIZE][MAX_SIZE];

  gather_offsets[threadIdx.x] = 0;
  int neighbor_size = 0;
  int neighbor_offset = 0;
  int scratch_offset = 0;
  int total_edges = 0;
  IndexT row_begin = 0;
  IndexT row_end = 0;

  AccType local_num = 0;
  if (pos < end - begin) {
    //emb_list.get_embedding(level, begin + pos, emb);
    emb_list.get_embedding(level, begin + pos, emb[tid]);
    auto vid = emb_list.get_vid(level, begin + pos);
    row_begin = graph.edge_begin(vid);
    row_end = graph.edge_end(vid);
    num_new_emb[pos] = 0;
    neighbor_offset = row_begin;
    neighbor_size = row_end - row_begin;
  }
  BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
  int done = 0;
  int neighbors_done = 0;
  while(total_edges > 0) {
    __syncthreads();
    int i;
    for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
      gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
      src[scratch_offset + i - done] = tid;
    }
    neighbors_done += i;
    scratch_offset += i;
    __syncthreads();
    if(tid < total_edges) {
      auto e = gather_offsets[tid];
      auto dst = graph.getEdgeDst(e);
      auto idx = src[tid];
      if (is_all_connected_dag(dst, emb[idx], level, graph)) {
        if (level < max_size-2) atomicAdd((AccType*)(num_new_emb+base_id+idx), 1);
        else local_num += 1;
      }
    }
    total_edges -= BLOCK_SIZE;
    done += BLOCK_SIZE;
  }
  AccType block_num = BlockReduce(reduce_storage).Sum(local_num);
  if (tid == 0) atomicAdd(total, block_num);
}


__global__ void extend_insert(size_t begin, size_t end, unsigned level, 
                              GraphGPU graph, EmbeddingList emb_list, emb_index_t *indices) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
#ifdef USE_SHM
  __shared__ IndexT emb[BLOCK_SIZE][MAX_SIZE];
#else
  IndexT emb[MAX_SIZE];
#endif
  if(pos < end - begin) {
#ifdef USE_SHM
    emb_list.get_embedding(level, begin + pos, emb[tid]);
#else
    emb_list.get_embedding(level, begin + pos, emb);
#endif
    IndexT vid = emb_list.get_vid(level, begin + pos);
    IndexT start = indices[pos];
    IndexT row_begin = graph.edge_begin(vid);
    IndexT row_end = graph.edge_end(vid);
    for (IndexT e = row_begin; e < row_end; e++) {
      IndexT dst = graph.getEdgeDst(e);
#ifdef USE_SHM
      if (is_all_connected_dag(dst, emb[tid], level, graph)) {
#else
      if (is_all_connected_dag(dst, emb, level, graph)) {
#endif
        emb_list.set_idx(level+1, start, begin + pos);
        emb_list.set_vid(level+1, start++, dst);
      }
    }
  }
}

#define N_CHUNK 1
void KclSolver(Graph &g, unsigned k, AccType &total) {
  //print_device_info(0);
  size_t m = g.num_vertices();
  size_t nnz = g.num_edges();
  int nthreads = BLOCK_SIZE;
  int nblocks = DIVIDE_INTO(m, nthreads);
  CUDA_Context_Mining cuda_ctx;
  cuda_ctx.hg = &g;
  cuda_ctx.build_graph_gpu();
  cuda_ctx.emb_list.init(nnz, k);
  init_gpu_dag<<<nblocks, nthreads>>>(m, cuda_ctx.gg, cuda_ctx.emb_list);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  AccType h_total = 0, *d_total;
  AccType zero = 0;
  size_t chunk_length = (nnz - 1) / N_CHUNK + 1;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  printf("Launching CUDA TC solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

  Timer t;
  t.Start();
  std::cout << "number of single-edge embeddings: " << nnz << "\n";
  for (size_t cid = 0; cid < N_CHUNK; cid ++) {
    size_t chunk_begin = cid * chunk_length;
    size_t chunk_end = std::min((cid+1) * chunk_length, nnz);
    size_t cur_size = chunk_end-chunk_begin;
    std::cout << "Processing the " << cid << " chunk of " << cur_size << " edges\n";

    unsigned level = 1;
    while (1) {
      emb_index_t *num_new_emb;
      size_t num_emb = cuda_ctx.emb_list.size();
      size_t begin = 0, end = num_emb;
      if (level == 1) { begin = chunk_begin; end = chunk_end; num_emb = end - begin; }
      std::cout << "\t number of embeddings in level " << level << ": " << num_emb << "\n";
      CUDA_SAFE_CALL(hipMalloc((void **)&num_new_emb, sizeof(emb_index_t) * (num_emb+1)));
      CUDA_SAFE_CALL(hipMemset(num_new_emb, 0, sizeof(emb_index_t) * (num_emb+1)));
      nblocks = (num_emb-1)/nthreads+1;
      CUDA_SAFE_CALL(hipMemcpy(d_total, &zero, sizeof(AccType), hipMemcpyHostToDevice));
      //std::cout << "\t Starting Extend_alloc ...\n";
      extend_alloc<<<nblocks, nthreads>>>(begin, end, level, k, cuda_ctx.gg, cuda_ctx.emb_list, num_new_emb, d_total);
      //extend_alloc_lb<<<nblocks, nthreads>>>(begin, end, level, k, cuda_ctx.gg, cuda_ctx.emb_list, (unsigned long long *)num_new_emb, d_total);
      CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
      total += h_total;
      CudaTest("solving extend alloc failed");
      //std::cout << "\t Extend_alloc Done\n";
      if (level == k-2) {
        CUDA_SAFE_CALL(hipFree(num_new_emb));
        break; 
      }
      emb_index_t *indices;
      CUDA_SAFE_CALL(hipMalloc((void **)&indices, sizeof(emb_index_t) * (num_emb+1)));
      thrust::exclusive_scan(thrust::device, num_new_emb, num_new_emb+num_emb+1, indices);
      //std::cout << "\t PrefixSum Done\n";
      CUDA_SAFE_CALL(hipFree(num_new_emb));
      size_t new_size;
      CUDA_SAFE_CALL(hipMemcpy(&new_size, &indices[num_emb], sizeof(unsigned), hipMemcpyDeviceToHost));
      std::cout << "\t number of new embeddings: " << new_size << "\n";
      cuda_ctx.emb_list.add_level(new_size);
      //std::cout << "\t Starting Extend_insert ...\n";
      extend_insert<<<nblocks, nthreads>>>(begin, end, level, cuda_ctx.gg, cuda_ctx.emb_list, indices);
      CudaTest("solving extend insert failed");
      //std::cout << "\t Extend_insert Done\n";
      CUDA_SAFE_CALL(hipFree(indices));
      level ++;
    }
    cuda_ctx.emb_list.reset_level();
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  printf("\truntime [cuda_base] = %f sec\n", t.Seconds());
  std::cout << "\n\ttotal_num_cliques = " << total << "\n\n";
  CUDA_SAFE_CALL(hipFree(d_total));
}

