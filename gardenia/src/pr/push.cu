#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "pr.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
//#define ENABLE_LB
#define PR_VARIANT "push"

typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void initialize(int m, ScoreT *sums) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) sums[id] = 0;
}

__global__ void push_base(int m, int *row_offsets, int *column_indices, ScoreT *scores, ScoreT *sums, int *processed) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		int degree = row_end - row_begin;
		ScoreT value = scores[src] / (ScoreT)degree;
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			atomicAdd(&sums[dst], value);
		}
	}
}

__device__ __forceinline__ void expandByCta(int m, const IndexT *row_offsets, const IndexT *column_indices, const ScoreT *scores, ScoreT *sums, int *processed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(id < m) {
		size = row_offsets[id+1] - row_offsets[id];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = id;
			processed[id] = 1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex+1];
		int neighbor_size = row_end - row_begin;
		ScoreT value = scores[sh_vertex] / (ScoreT)neighbor_size;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[edge];
				atomicAdd(&sums[dst], value);
			}
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, const int *row_offsets, const int *column_indices, const ScoreT *scores, ScoreT *sums, int *processed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	if(id < m && !processed[id]) {
		size = row_offsets[id+1] - row_offsets[id];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = id;
			processed[id] = 1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		ScoreT value = scores[winner] / (ScoreT)neighbor_size;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[edge];
				atomicAdd(&sums[dst], value);
			}
		}
	}
}

__global__ void push_lb(int m, IndexT *row_offsets, IndexT *column_indices, ScoreT *scores, ScoreT *sums, int *processed) {
	expandByCta(m, row_offsets, column_indices, scores, sums, processed);
	expandByWarp(m, row_offsets, column_indices, scores, sums, processed);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int src = tid;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int src_idx[BLOCK_SIZE];
	__shared__ ScoreT values[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	src_idx[tx] = 0;
	values[tx] = 0;
	__syncthreads();

	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (tid < m && !processed[tid]) {
		neighbor_offset = row_offsets[tid];
		neighbor_size = row_offsets[tid+1] - neighbor_offset;
		values[tx] = scores[src] / (ScoreT)neighbor_size;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	
	int done = 0;
	int neighbors_done = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			src_idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int dst = column_indices[edge];
			atomicAdd(&sums[dst], values[src_idx[tx]]);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void l1norm(int m, ScoreT *scores, ScoreT *sums, float *diff, ScoreT base_score) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(u < m) {
		ScoreT new_score = base_score + kDamp * sums[u];
		local_diff += fabs(new_score - scores[u]);
		scores[u] = new_score;
		sums[u] = 0;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degrees, ScoreT *h_scores) {
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	ScoreT *d_sums;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_sums, m * sizeof(ScoreT)));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	int *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(int)));

	int iter = 0;
	const ScoreT base_score = (1.0f - kDamp) / m;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, d_sums);
	CudaTest("init failed");
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(int)));
#ifdef ENABLE_LB
		push_lb <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_scores, d_sums, d_processed);
#else
		push_base <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_scores, d_sums, d_processed);
#endif
		CudaTest("solving kernel push failed");
		l1norm <<<nblocks, nthreads>>> (m, d_scores, d_sums, d_diff, base_score);
		CudaTest("solving kernel reduce failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		//printf("iteration=%d, diff=%f\n", iter, h_diff);
		printf(" %2d    %lf\n", iter, h_diff);
		//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_sums));
	CUDA_SAFE_CALL(hipFree(d_processed));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
