#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "pr.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#define PR_VARIANT "pull_lb"

typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void contrib(int m, ScoreT *scores, int *degree, ScoreT *outgoing_contrib) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	if (u < m) outgoing_contrib[u] = scores[u] / degree[u];
}

__global__ void l1norm(int m, ScoreT *scores, ScoreT *sums, float *diff, ScoreT base_score) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(u < m) {
		ScoreT new_score = base_score + kDamp * sums[u];
		local_diff += fabs(new_score - scores[u]);
		scores[u] = new_score;
		sums[u] = 0;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

__device__ __forceinline__ void expandByCta(int m, const IndexT *row_offsets, const IndexT *column_indices, ScoreT *sums, const ScoreT *outgoing_contrib, bool *processed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	int dst = id;
	if(dst < m) size = row_offsets[dst+1] - row_offsets[dst];
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = dst;
			processed[dst] = 1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		ScoreT sum = 0;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				//int src = column_indices[edge];
				int src = __ldg(column_indices+edge);
				//sum += outgoing_contrib[src];
				sum += __ldg(outgoing_contrib+src);
			}
		}
		ScoreT block_sum = BlockReduce(temp_storage).Sum(sum);
		if(threadIdx.x == 0) sums[sh_vertex] = block_sum;
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, const IndexT *row_offsets, const IndexT *column_indices, ScoreT *sums, const ScoreT *outgoing_contrib, bool *processed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	__shared__ ScoreT sdata[BLOCK_SIZE + 16];
	owner[warp_id] = -1;
	int size = 0;
	int dst = id;
	if(dst < m && !processed[dst]) {
		size = row_offsets[dst+1] - row_offsets[dst];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = dst;
			processed[dst] = 1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		ScoreT sum = 0;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				//int src = column_indices[edge];
				int src = __ldg(column_indices+edge);
				//sum += outgoing_contrib[src];
				sum += __ldg(outgoing_contrib+src);
			}
		}
		sdata[threadIdx.x] = sum; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
		if(lane_id == 0) sums[winner] += sdata[threadIdx.x];
	}
}

__global__ void pull_base(int m, const IndexT *row_offsets, const IndexT *column_indices, ScoreT *sums, const ScoreT *outgoing_contrib, bool *processed) {
	expandByCta(m, row_offsets, column_indices, sums, outgoing_contrib, processed);
	expandByWarp(m, row_offsets, column_indices, sums, outgoing_contrib, processed);
	int dst = blockIdx.x * blockDim.x + threadIdx.x;
	if (dst < m && !processed[dst]) {
		IndexT row_begin = row_offsets[dst];
		IndexT row_end = row_offsets[dst+1];
		ScoreT incoming_total = 0;
		for (IndexT offset = row_begin; offset < row_end; ++ offset) {
			//IndexT src = column_indices[offset];
			IndexT src = __ldg(column_indices+offset);
			//incoming_total += outgoing_contrib[src];
			incoming_total += __ldg(outgoing_contrib+src);
		}
		sums[dst] = incoming_total;
	}
}

__global__ void pull_lb(int m, const IndexT *row_offsets, const IndexT *column_indices, ScoreT *sums, const ScoreT *outgoing_contrib, bool *processed) {
	expandByCta(m, row_offsets, column_indices, sums, outgoing_contrib, processed);
	expandByWarp(m, row_offsets, column_indices, sums, outgoing_contrib, processed);
	int dst = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int dst_idx[BLOCK_SIZE];
	__shared__ ScoreT sh_total[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	dst_idx[tx] = 0;
	sh_total[tx] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (dst < m && !processed[dst]) {
		neighbor_offset = row_offsets[dst];
		neighbor_size = row_offsets[dst+1] - neighbor_offset;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			dst_idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int offset = gather_offsets[tx];
			int src = column_indices[offset];
			atomicAdd(&sh_total[dst_idx[tx]], __ldg(outgoing_contrib+src));
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
	__syncthreads();
	if (dst < m && !processed[dst])
		sums[dst] = sh_total[tx];
}

void PRSolver(int m, int nnz, IndexT *in_row_offsets, IndexT *in_column_indices, IndexT *out_row_offsets, IndexT *out_column_indices, int *degrees, ScoreT *scores) {
	//print_device_info(0);
	IndexT *d_row_offsets, *d_column_indices;
	int *d_degrees;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(IndexT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(IndexT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degrees, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, in_row_offsets, (m + 1) * sizeof(IndexT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, in_column_indices, nnz * sizeof(IndexT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degrees, degrees, m * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores, *d_sums, *d_contrib;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_sums, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	bool *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(bool)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	const ScoreT base_score = (1.0f - kDamp) / m;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		contrib<<<nblocks, nthreads>>>(m, d_scores, d_degrees, d_contrib);
		CudaTest("solving kernel contrib failed");
		CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(bool)));
		pull_base<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_sums, d_contrib, d_processed);
		l1norm<<<nblocks, nthreads>>> (m, d_scores, d_sums, d_diff, base_score);
		CudaTest("solving kernel pull failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		printf(" %2d    %f\n", iter, h_diff);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degrees));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_sums));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
