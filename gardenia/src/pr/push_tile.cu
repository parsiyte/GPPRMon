#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "pr.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#define GPU_SEGMENTING
#include "segmenting.h"
#define ENABLE_LB
#define PR_VARIANT "push_tile"

typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void initialize(int m, ScoreT *sums) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) sums[id] = 0;
}

__global__ void contrib(int m, ScoreT *scores, int *degrees, ScoreT *outgoing_contrib) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	if (u < m) outgoing_contrib[u] = scores[u] / degrees[u];
}

__global__ void push_base(int m, const IndexT *row_offsets, const IndexT *column_indices, const IndexT *idx_map, const ScoreT *contrib, ScoreT *sums, int *processed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < m) {
		int row_begin = row_offsets[id];
		int row_end = row_offsets[id+1];
		int src = idx_map[id];
		ScoreT value = contrib[src];
		//ScoreT value = __ldg(contrib+src);
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			//int dst = __ldg(column_indices+offset);
			atomicAdd(&sums[dst], value);
		}
	}
}

__device__ __forceinline__ void expandByCta(int m, const IndexT *row_offsets, const IndexT *column_indices, const IndexT *idx_map, const ScoreT *contrib, ScoreT *sums, int *processed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(id < m) {
		size = row_offsets[id+1] - row_offsets[id];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = id;
			processed[id] = 1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex+1];
		int src = idx_map[sh_vertex];
		ScoreT value = contrib[src];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[edge];
				atomicAdd(&sums[dst], value);
			}
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, const IndexT *row_offsets, const IndexT *column_indices, const IndexT *idx_map, const ScoreT *contrib, ScoreT *sums, int *processed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	if(id < m && !processed[id]) {
		size = row_offsets[id+1] - row_offsets[id];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = id;
			processed[id] = 1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int src = idx_map[winner];
		ScoreT value = contrib[src];
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[edge];
				atomicAdd(&sums[dst], value);
			}
		}
	}
}

__global__ void push_lb(int m, const IndexT *row_offsets, const IndexT *column_indices, const IndexT *idx_map, const ScoreT *contrib, ScoreT *sums, int *processed) {
	expandByCta(m, row_offsets, column_indices, idx_map, contrib, sums, processed);
	//expandByWarp(m, row_offsets, column_indices, idx_map, contrib, sums, processed);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int src_idx[BLOCK_SIZE];
	__shared__ ScoreT values[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	src_idx[tx] = 0;
	values[tx] = 0;
	__syncthreads();

	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (tid < m && !processed[tid]) {
		neighbor_offset = row_offsets[tid];
		neighbor_size = row_offsets[tid+1] - neighbor_offset;
		int src = idx_map[tid];
		values[tx] = contrib[src];
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			src_idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int dst = column_indices[edge];
			atomicAdd(&sums[dst], values[src_idx[tx]]);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void l1norm(int m, ScoreT *scores, ScoreT *sums, float *diff, ScoreT base_score) {
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(u < m) {
		ScoreT new_score = base_score + kDamp * sums[u];
		local_diff += fabs(new_score - scores[u]);
		scores[u] = new_score;
		sums[u] = 0;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, IndexT *in_row_offsets, IndexT *in_column_indices, IndexT *h_row_offsets, IndexT *h_column_indices, int *h_degrees, ScoreT *h_scores) {
	segmenting(m, h_row_offsets, h_column_indices, NULL);
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	int *d_degrees;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degrees, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_degrees, h_degrees, m * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_sums, *d_contrib;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_sums, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));
	int *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(int)));

	int num_subgraphs = (m - 1) / SUBGRAPH_SIZE + 1;
	vector<IndexT *> d_rowptr_blocked(num_subgraphs), d_colidx_blocked(num_subgraphs);
	IndexT ** d_idx_map = (IndexT**)malloc(num_subgraphs * sizeof(IndexT*));

	printf("copy host data to device\n");
	for (int bid = 0; bid < num_subgraphs; bid ++) {
		CUDA_SAFE_CALL(hipMalloc((void **)&d_rowptr_blocked[bid], (ms_of_subgraphs[bid] + 1) * sizeof(IndexT)));
		CUDA_SAFE_CALL(hipMalloc((void **)&d_colidx_blocked[bid], nnzs_of_subgraphs[bid] * sizeof(IndexT)));
		CUDA_SAFE_CALL(hipMalloc((void **)&d_idx_map[bid], ms_of_subgraphs[bid] * sizeof(IndexT)));
		CUDA_SAFE_CALL(hipMemcpy(d_rowptr_blocked[bid], rowptr_blocked[bid], (ms_of_subgraphs[bid] + 1) * sizeof(IndexT), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_colidx_blocked[bid], colidx_blocked[bid], nnzs_of_subgraphs[bid] * sizeof(IndexT), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_idx_map[bid], idx_map[bid], ms_of_subgraphs[bid] * sizeof(IndexT), hipMemcpyHostToDevice));
	}

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	const ScoreT base_score = (1.0f - kDamp) / m;
	initialize <<<nblocks, nthreads>>> (m, d_sums);
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		contrib <<<nblocks, nthreads>>>(m, d_scores, d_degrees, d_contrib);
		CudaTest("solving kernel contrib failed");
		for (int bid = 0; bid < num_subgraphs; bid ++) {
			//Timer tt;
			//tt.Start();
			int msub = ms_of_subgraphs[bid];
			//int nnz = nnzs_of_subgraphs[bid];
			int bblocks = (msub - 1) / nthreads + 1;
			CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(int)));
#ifdef ENABLE_LB
			push_lb<<<bblocks, nthreads>>>(msub, d_rowptr_blocked[bid], d_colidx_blocked[bid], d_idx_map[bid], d_contrib, d_sums, d_processed);
#else
			push_base<<<bblocks, nthreads>>>(msub, d_rowptr_blocked[bid], d_colidx_blocked[bid], d_idx_map[bid], d_contrib, d_sums, d_processed);
#endif
			CUDA_SAFE_CALL(hipDeviceSynchronize());
			//tt.Stop();
			//if(iter == 1) printf("\truntime subgraph[%d] = %f ms.\n", bid, tt.Millisecs());
		}
		CudaTest("solving kernel push failed");
		l1norm <<<nblocks, nthreads>>> (m, d_scores, d_sums, d_diff, base_score);
		CudaTest("solving kernel l1norm failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		printf(" %2d    %lf\n", iter, h_diff);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_degrees));
	CUDA_SAFE_CALL(hipFree(d_sums));
	CUDA_SAFE_CALL(hipFree(d_processed));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
