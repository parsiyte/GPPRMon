#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Authors: Xuhao Chen <cxh@mit.edu>
#include <hipcub/hipcub.hpp>
#include "vc.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "worklistc.h"

__device__ __forceinline__ void assignColor(unsigned int *forbiddenColors, int *colors, int node) {
	int i;
/*
	int c = 32;
	for (i = 0; i < MAXCOLOR/32; i++) {
		if (forbiddenColors[i] != 0) {
			forbiddenColors[i] &= -(signed)forbiddenColors[i];
			if (forbiddenColors[i]) c--;
			if (forbiddenColors[i] & 0x0000ffff) c -= 16;
	        	if (forbiddenColors[i] & 0x00ff00ff) c -= 8;
        		if (forbiddenColors[i] & 0x0f0f0f0f) c -= 4;
		        if (forbiddenColors[i] & 0x33333333) c -= 2;
        		if (forbiddenColors[i] & 0x55555555) c -= 1;
			colors[node] = c;
			break;
		}
		else
			c += 32;
	}
//*/
///*
	for (i = 0; i < MAXCOLOR/32; i++) {
		int pos = __ffs(forbiddenColors[i]);
		if(pos) {
			colors[node] = i * 32 + pos - 1;
			break;
		}
	}
	assert(i < MAXCOLOR/32);
//*/
}

__global__ void first_fit(int m, uint64_t *row_offsets, 
                          int *column_indices, 
                          Worklist2 inwl, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned forbiddenColors[MAXCOLOR/32+1];
	int vertex;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int j = 0; j < MAXCOLOR/32; j++)
			forbiddenColors[j] = 0xffffffff;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			int color = colors[neighbor];
			forbiddenColors[color / 32] &= ~(1 << (color % 32));
		}
		assignColor(forbiddenColors, colors, vertex);
	}
}

__global__ void conflict_resolve(int m, uint64_t *row_offsets, 
                                 int *column_indices, 
                                 Worklist2 inwl, 
                                 Worklist2 outwl, 
                                 int *colors) {
	//typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	int conflicted = 0;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			if (colors[vertex] == colors[neighbor] && vertex < neighbor) {
				conflicted = 1;
				colors[vertex] = MAXCOLOR;
				break;
			}
		}
	}
	//outwl.push_1item<BlockScan>(conflicted, vertex, BLOCK_SIZE);
	if(conflicted) outwl.push(vertex);
}

int VCSolver(Graph &g, int *colors) {
  auto m = g.V();
  auto nnz = g.E();
  auto h_row_offsets = g.out_rowptr();
  auto h_column_indices = g.out_colidx();	
  //print_device_info(0);
  uint64_t *d_row_offsets;
  VertexId *d_column_indices;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(VertexId)));
  CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
 
	int *d_colors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colors, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_colors, colors, m * sizeof(int), hipMemcpyHostToDevice));

	int nitems = m;
	int num_colors = 0, iter = 0;
	Worklist2 inwl(m), outwl(m);
	Worklist2 *inwlptr = &inwl, *outwlptr = &outwl;
	for(int i = 0; i < m; i ++) inwl.h_queue[i] = i;
	inwl.set_index(m);
	CUDA_SAFE_CALL(hipMemcpy(inwl.d_queue, inwl.h_queue, m * sizeof(int), hipMemcpyHostToDevice));
	//thrust::sequence(thrust::device, inwl.d_queue, inwl.d_queue + m);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("Launching CUDA VC solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	Timer t;
	t.Start();
	while (nitems > 0) {
		iter ++;
		int nblocks = (nitems - 1) / BLOCK_SIZE + 1;
		first_fit<<<nblocks, BLOCK_SIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, d_colors);
		conflict_resolve<<<nblocks, BLOCK_SIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, *outwlptr, d_colors);
		nitems = outwlptr->nitems();
		Worklist2 * tmp = inwlptr;
		inwlptr = outwlptr;
		outwlptr = tmp;
		outwlptr->reset();
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	CUDA_SAFE_CALL(hipMemcpy(colors, d_colors, m * sizeof(int), hipMemcpyDeviceToHost));
	#pragma omp parallel for reduction(max : num_colors)
	for (int n = 0; n < m; n ++)
		num_colors = max(num_colors, colors[n]);
	num_colors ++;
    printf("\titerations = %d.\n", iter);
    printf("\truntime[cuda_linear_bitset] = %f ms, num_colors = %d.\n", t.Millisecs(), num_colors);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_colors));
	return num_colors;
}

