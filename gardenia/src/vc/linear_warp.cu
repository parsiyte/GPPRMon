#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#define VC_VARIANT "linear_warp"
#include <hipcub/hipcub.hpp>
#include "vc.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "worklistc.h"

__device__ __forceinline__ void assignColor(unsigned int *forbiddenColors, int *colors, int node) {
	int i;
	for (i = 0; i < MAXCOLOR/32; i++) {
		int pos = __ffs(forbiddenColors[i]);
		if(pos) {
			colors[node] = i * 32 + pos - 1;
			break;
		}
	}
	assert(i < MAXCOLOR/32);
}

__global__ void first_fit(int *row_offsets, int *column_indices, Worklist2 inwl, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned forbiddenColors[MAXCOLOR/32+1];
	int vertex;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int j = 0; j < MAXCOLOR/32; j++)
			forbiddenColors[j] = 0xffffffff;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			int color = colors[neighbor];
			forbiddenColors[color / 32] &= ~(1 << (color % 32));
		}
		assignColor(forbiddenColors, colors, vertex);
	}
}

__global__ void conflict_resolve(int nitems, int *row_offsets, int *column_indices, Worklist2 inwl, Worklist2 outwl, int *colors) {
/*
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	int conflicted = 0;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			if (colors[vertex] == colors[neighbor] && vertex < neighbor) {
				conflicted = 1;
				colors[vertex] = MAXCOLOR;
				break;
			}
		}
	}
	if(conflicted) outwl.push(vertex);
//*/
///*
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
	__shared__ bool conflicted[BLOCK_SIZE/WARP_SIZE];

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int index = warp_id; index < nitems; index += num_warps) {
		int src;
		inwl.pop_id(index, src);
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = row_offsets[src + thread_lane];
		const int row_start = ptrs[warp_lane][0];
		const int row_end   = ptrs[warp_lane][1];
	
		if (thread_lane == 0) conflicted[warp_lane] = false;
		__syncthreads();
		bool is_conflicted = false;
		for(int offset = row_start + thread_lane; offset < row_end; offset += WARP_SIZE) {
			int dst = column_indices[offset];
			if(src < dst && colors[src] == colors[dst]) is_conflicted = true;
			if(__any_sync(0xFFFFFFFF, is_conflicted)) { conflicted[warp_lane] = true; break; }
		}
		if (thread_lane == 0 && conflicted[warp_lane]) {
			colors[src] = MAXCOLOR;
			outwl.push(src);
		}
	}
	//*/
}

int VCSolver(int m, int nnz, int *row_offsets, int *column_indices, int *colors) {
	int num_colors = 0, iter = 0;
	int *d_row_offsets, *d_column_indices, *d_colors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colors, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_colors, colors, m * sizeof(int), hipMemcpyHostToDevice));

	int nitems = m;
	Worklist2 inwl(m), outwl(m);
	Worklist2 *inwlptr = &inwl, *outwlptr = &outwl;
	for(int i = 0; i < m; i ++) inwl.h_queue[i] = i;
	inwl.set_index(m);
	CUDA_SAFE_CALL(hipMemcpy(inwl.d_queue, inwl.h_queue, m * sizeof(int), hipMemcpyHostToDevice));
	//thrust::sequence(thrust::device, inwl.d_queue, inwl.d_queue + m);
	const int nthreads = BLOCK_SIZE;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	const int nSM = deviceProp.multiProcessorCount;
	const int max_blocks_per_SM = maximum_residency(conflict_resolve, nthreads, 0);
	const int max_blocks = max_blocks_per_SM * nSM;
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("Launching CUDA VC solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	Timer t;
	t.Start();
	while (nitems > 0) {
		iter ++;
		const int mblocks = (nitems - 1) / nthreads + 1;
		first_fit<<<mblocks, nthreads>>>(d_row_offsets, d_column_indices, *inwlptr, d_colors);
		const int nblocks = std::min(max_blocks, DIVIDE_INTO(nitems, WARPS_PER_BLOCK));
		conflict_resolve<<<nblocks, nthreads>>>(nitems, d_row_offsets, d_column_indices, *inwlptr, *outwlptr, d_colors);
		nitems = outwlptr->nitems();
		Worklist2 * tmp = inwlptr;
		inwlptr = outwlptr;
		outwlptr = tmp;
		outwlptr->reset();
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	CUDA_SAFE_CALL(hipMemcpy(colors, d_colors, m * sizeof(int), hipMemcpyDeviceToHost));
	#pragma omp parallel for reduction(max : num_colors)
	for (int n = 0; n < m; n ++)
		num_colors = max(num_colors, colors[n]);
	num_colors ++;
    printf("\titerations = %d.\n", iter);
    printf("\truntime[%s] = %f ms, num_colors = %d.\n", VC_VARIANT, t.Millisecs(), num_colors);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_colors));
	return num_colors;
}

