#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#define VC_VARIANT "topo_bitset"
#include "vc.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"

__device__ __forceinline__ void assignColor(unsigned *forbiddenColors, int *colors, int node) {
	int i;
	for (i = 0; i < MAXCOLOR/32; i++) {
		int pos = __ffs(forbiddenColors[i]);
		if(pos) {
			colors[node] = i * 32 + pos - 1;
			break;
		}
	}
	assert(i < MAXCOLOR/32);
}

__global__ void first_fit(int m, int *row_offsets, int *column_indices, int *colors, bool *changed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
	unsigned forbiddenColors[MAXCOLOR/32+1];
	if (colors[id] == MAXCOLOR) {
		int row_begin = row_offsets[id];
		int row_end = row_offsets[id+1];
		for (int j = 0; j < MAXCOLOR/32; j++)
			forbiddenColors[j] = 0xffffffff;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			int color = colors[neighbor];
			forbiddenColors[color / 32] &= ~(1 << (color % 32));
		}
		assignColor(forbiddenColors, colors, id);
		*changed = true;
	}
}

__global__ void conflict_resolve(int m, int *row_offsets, int *column_indices, int *colors, bool *colored) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m && !colored[id]) {
		int row_begin = row_offsets[id];
		int row_end = row_offsets[id + 1];
		colored[id] = true;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			if (id < neighbor && colors[id] == colors[neighbor]) {
				colors[id] = MAXCOLOR;
				colored[id] = false;
				break;
			}
		}
	}
}

int VCSolver(int m, int nnz, int *row_offsets, int *column_indices, int *colors) {
	//print_device_info(0);
	int *d_row_offsets, *d_column_indices, *d_colors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colors, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_colors, colors, m * sizeof(int), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_colored;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colored, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_colored, 0, m * sizeof(bool)));

	int num_colors = 0, iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA VC solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();	
	do {
		iter ++;
		//printf("iteration=%d\n", iter);
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		first_fit<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_colors, d_changed);
		CudaTest("first_fit failed");
		conflict_resolve<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_colors, d_colored);
		CudaTest("conflict_resolve failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	CUDA_SAFE_CALL(hipMemcpy(colors, d_colors, m * sizeof(int), hipMemcpyDeviceToHost));
	#pragma omp parallel for reduction(max : num_colors)
	for (int n = 0; n < m; n ++)
		num_colors = max(num_colors, colors[n]);
	num_colors ++;	
	printf("\titerations = %d.\n", iter);
	printf("\truntime[%s] = %f ms, num_colors = %d.\n", VC_VARIANT, t.Millisecs(), num_colors);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_colors));
	return num_colors;
}
