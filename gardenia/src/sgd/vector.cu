#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>

/*
Gardenia Benchmark Suite
Kernel: Stochastic Gradient Descent (SGD)
Author: Xuhao Chen
*/
#define SGD_VARIANT "vector"
#include "sgd.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
#define USE_SHFL

#if __CUDACC_VER_MAJOR__ >= 9
#define SHFL_DOWN(a,b) __shfl_down_sync(0xFFFFFFFF,a,b)
#define SHFL(a,b) __shfl_sync(0xFFFFFFFF,a,b)
#else
#define SHFL_DOWN(a,b) __shfl_down(a,b)
#define SHFL(a,b) __shfl(a,b)
#endif
typedef hipcub::BlockReduce<ScoreT, BLOCK_SIZE> BlockReduce;

__global__ void update(int m, int n, int *row_offsets, int *column_indices, ScoreT *rating, LatentT *user_lv, LatentT *item_lv, ScoreT lambda, ScoreT step, int *ordering, ScoreT *squared_errors) {
#ifndef USE_SHFL
	__shared__ ScoreT sdata[BLOCK_SIZE + 16];                       // padded to avoid reduction ifs
#endif
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int index = warp_id; index < m; index += num_warps) {
		//int user_id = ordering[index];
		int user_id = index;
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = row_offsets[user_id + thread_lane];
		const int row_begin = ptrs[warp_lane][0];                   //same as: row_begin = row_offsets[row];
		const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = row_offsets[row+1];
		//for(int offset = row_begin + thread_lane; offset < row_end; offset += WARP_SIZE) {
		for(int offset = row_begin; offset < row_end; offset ++) {
			int item_id = column_indices[offset];
			int base_p = user_id * K;
			int base_q = item_id * K;
			LatentT temp_p[K/WARP_SIZE + 1];
			LatentT temp_q[K/WARP_SIZE + 1];
			ScoreT estimate = 0;
			for (int i = 0; i < K; i += WARP_SIZE) {
				int j = i/WARP_SIZE;
				temp_p[j] = user_lv[base_p+thread_lane+i];
				temp_q[j] = item_lv[base_q+thread_lane+i];
				estimate += temp_p[j] * temp_q[j];
			}
#ifdef USE_SHFL
			estimate += SHFL_DOWN(estimate, 16);
			estimate += SHFL_DOWN(estimate, 8);
			estimate += SHFL_DOWN(estimate, 4);
			estimate += SHFL_DOWN(estimate, 2);
			estimate += SHFL_DOWN(estimate, 1);
			estimate = SHFL(estimate, 0);
#else
			sdata[threadIdx.x] = estimate; __syncthreads();
			sdata[threadIdx.x] = estimate = estimate + sdata[threadIdx.x + 16]; __syncthreads();
			sdata[threadIdx.x] = estimate = estimate + sdata[threadIdx.x +  8]; __syncthreads();
			sdata[threadIdx.x] = estimate = estimate + sdata[threadIdx.x +  4]; __syncthreads();
			sdata[threadIdx.x] = estimate = estimate + sdata[threadIdx.x +  2]; __syncthreads();
			sdata[threadIdx.x] = estimate = estimate + sdata[threadIdx.x +  1]; __syncthreads();
			estimate = sdata[warp_lane*WARP_SIZE];
#endif
			ScoreT delta = rating[offset] - estimate;
			if (thread_lane == 0) squared_errors[user_id] += delta * delta;
			for (int i = 0; i < K; i += WARP_SIZE) {
				int j = i/WARP_SIZE;
				LatentT new_user_feature = temp_p[j] + step * (-lambda * temp_p[j] + temp_q[j] * delta);
				LatentT new_item_feature = temp_q[j] + step * (-lambda * temp_q[j] + temp_p[j] * delta);
				user_lv[base_p+thread_lane+i] = new_user_feature;
				item_lv[base_q+thread_lane+i] = new_item_feature;
			}
		}
	}
}

__global__ void rmse(int m, ScoreT *squared_errors, ScoreT *total_error) {
	int uid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	ScoreT local_error = 0.0;
	if(uid < m) local_error = squared_errors[uid];
	ScoreT block_sum = BlockReduce(temp_storage).Sum(local_error);
	if(threadIdx.x == 0) atomicAdd(total_error, block_sum);
}

void SGDSolver(int num_users, int num_items, int nnz, int *h_row_offsets, int *h_column_indices, ScoreT *h_rating, LatentT *h_user_lv, LatentT *h_item_lv, int *h_ordering) {
	//print_device_info(0);
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (num_users + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (num_users + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_rating;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_rating, nnz * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_rating, h_rating, nnz * sizeof(ScoreT), hipMemcpyHostToDevice));
	int *d_ordering;
	//CUDA_SAFE_CALL(hipMalloc((void **)&d_ordering, num_users * sizeof(int)));
	//CUDA_SAFE_CALL(hipMemcpy(d_ordering, h_ordering, num_users * sizeof(int), hipMemcpyHostToDevice));

	LatentT *d_user_lv, *d_item_lv;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_user_lv, num_users * K * sizeof(LatentT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_item_lv, num_items * K * sizeof(LatentT)));
	CUDA_SAFE_CALL(hipMemcpy(d_user_lv, h_user_lv, num_users * K * sizeof(LatentT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_item_lv, h_item_lv, num_items * K * sizeof(LatentT), hipMemcpyHostToDevice));
	ScoreT h_error, *d_error, *squared_errors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_error, sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMalloc((void **)&squared_errors, num_users * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemset(d_error, 0, sizeof(ScoreT)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (num_users - 1) / WARPS_PER_BLOCK + 1;
	printf("Launching CUDA SGD solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	Timer t;
	t.Start();
	do {
		++iter;
		h_error = 0.0;
		CUDA_SAFE_CALL(hipMemset(squared_errors, 0, num_users * sizeof(ScoreT)));
		CUDA_SAFE_CALL(hipMemcpy(d_error, &h_error, sizeof(ScoreT), hipMemcpyHostToDevice));
		update<<<nblocks, nthreads>>>(num_users, num_items, d_row_offsets, d_column_indices, d_rating, d_user_lv, d_item_lv, lambda, step, d_ordering, squared_errors);
		CudaTest("solving kernel update failed");
		rmse<<<nblocks, nthreads>>>(num_users, squared_errors, d_error);
		CudaTest("solving kernel rmse failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_error, d_error, sizeof(ScoreT), hipMemcpyDeviceToHost));
		//printf("h_error=%f\n", h_error);
		printf("iteration %d: RMSE error = %f\n", iter, sqrt(h_error/nnz));
		//CUDA_SAFE_CALL(hipMemcpy(h_user_lv, d_user_lv, num_users * K * sizeof(LatentT), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(h_item_lv, d_item_lv, num_items * K * sizeof(LatentT), hipMemcpyDeviceToHost));
		//print_latent_vector(num_users, num_items, h_user_lv, h_item_lv);
	} while (iter < max_iters && h_error > epsilon);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", SGD_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_user_lv, d_user_lv, num_users * K * sizeof(LatentT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_item_lv, d_item_lv, num_items * K * sizeof(LatentT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_rating));
	CUDA_SAFE_CALL(hipFree(d_user_lv));
	CUDA_SAFE_CALL(hipFree(d_item_lv));
	CUDA_SAFE_CALL(hipFree(d_error));
	CUDA_SAFE_CALL(hipFree(squared_errors));
}

