#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define SSSP_VARIANT "topo_base"
#include "sssp.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"

//Naive CUDA implementation of the Bellman-Ford algorithm for SSSP
__global__ void initialize(int m, int source, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		expanded[id] = false;
		if(id == source) visited[id] = true;
		else visited[id] = false;
	}
}

/**
 * @brief naive Bellman_Ford SSSP kernel entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] d_row_offsets     Device pointer of VertexId to the row offsets queue
 * @param[in] d_column_indices  Device pointer of VertexId to the column indices queue
 * @param[in] d_weight          Device pointer of DistT to the edge weight queue
 * @param[out]d_dist            Device pointer of DistT to the distance queue
 * @param[in] d_in_queue        Device pointer of VertexId to the incoming frontier queue
 * @param[out]d_out_queue       Device pointer of VertexId to the outgoing frontier queue
 */
__global__ void bellman_ford(int m, int *row_offsets, int *column_indices, DistT *weight, DistT *dist, bool *changed, bool *visited, bool *expanded, int *num_frontier) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		//atomicAdd(num_frontier, 1);
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			DistT old_dist = dist[dst]; 
			DistT new_dist = dist[src] + weight[offset];
			if (new_dist < old_dist) {
				if (atomicMin(&dist[dst], new_dist) > new_dist) {
					if(expanded[dst]) expanded[dst] = false;
					*changed = true;
				}
			}
		}
	}
}

__global__ void update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

/**
 * @brief naive topology-driven mapping GPU SSSP entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] h_row_offsets     Host pointer of VertexId to the row offsets queue
 * @param[in] h_column_indices  Host pointer of VertexId to the column indices queue
 * @param[in] h_weight          Host pointer of DistT to the edge weight queue
 * @param[out]h_dist            Host pointer of DistT to the distance queue
 */
void SSSPSolver(int m, int nnz, int source, int *h_row_offsets, int *h_column_indices, DistT *h_weight, DistT *h_dist, int delta) {
	//print_device_info(0);
	DistT zero = 0;
	bool one = 1;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT *d_weight;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_weight, nnz * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_weight, h_weight, nnz * sizeof(DistT), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));
	int *d_num_frontier;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_num_frontier, sizeof(int)));
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	int iter = 0;
	//int h_num_frontier = 1;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SSSP solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_num_frontier, &zero, sizeof(int), hipMemcpyHostToDevice));
		bellman_ford<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_weight, d_dist, d_changed, d_visited, d_expanded, d_num_frontier);
		update<<<nblocks, nthreads>>>(m, d_dist, d_visited);
		CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_num_frontier, d_num_frontier, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration %d: num_frontier = %d\n", iter, h_num_frontier);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", SSSP_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_weight));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	CUDA_SAFE_CALL(hipFree(d_num_frontier));
	return;
}
