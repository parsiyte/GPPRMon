#include "hip/hip_runtime.h"
// Copyright 2020, MIT
// Authors: Xuhao Chen <cxh@mit.edu>
#include <hipcub/hipcub.hpp>
#include "timer.h"
#include "graph_gpu.h"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

// warp-wise edge-parallel: each warp takes one edge
__global__ void warp_edge(eidType ne, GraphGPU g, AccType *total) {
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x; // global thread index
  int warp_id   = thread_id   / WARP_SIZE;               // global warp index
  int num_warps = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;  // total number of active warps
  AccType count = 0;
  for (eidType eid = warp_id; eid < ne; eid += num_warps) {
    auto v = g.get_src(eid);
    auto u = g.get_dst(eid);
    count += g.warp_intersect_cache(v, u);
  }
  AccType block_num = BlockReduce(temp_storage).Sum(count);
  if (threadIdx.x == 0) atomicAdd(total, block_num);
}

void TCSolver(Graph &g, uint64_t &total) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  auto nnz = gg.init_edgelist(g);
  std::cout << "Edge parallel: edgelist size = " << nnz << "\n";
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_edge, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  //size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  //nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
 
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
  warp_edge<<<nblocks, nthreads>>>(ne, gg, d_total);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [gpu_base] = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(nnz) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

