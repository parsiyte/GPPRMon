#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "bfs.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#define BFS_VARIANT "topo_base"

__global__ void bfs_step(int m, const IndexT *row_offsets, const IndexT *column_indices, int *front, DistT *depths, int depth) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && front[src]) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if (depths[dst] == MYINFINITY) {
				//atomicMin(&depths[dst], depth);
				depths[dst] = depth;
			}
		}
	}
}

__global__ void update(int m, DistT *depths, bool *visited, int *front, bool *changed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(depths[id] != MYINFINITY && !visited[id]) {
			visited[id] = true;
			front[id] = 1;
			*changed = true;
		}
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *in_degree, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	bool one = 1;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	int *d_front;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_front, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_front, 0, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(&d_front[source], &one, sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		bfs_step <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_front, d_dist, iter);
		CudaTest("solving bfs_step failed");
		CUDA_SAFE_CALL(hipMemset(d_front, 0, m * sizeof(int)));
		update <<<nblocks, nthreads>>> (m, d_dist, d_visited, d_front, d_changed);
		CudaTest("solving update failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_front));
	CUDA_SAFE_CALL(hipFree(d_changed));
	return;
}
