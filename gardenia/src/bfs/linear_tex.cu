#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "linear_base"
#include "bfs.h"
#include "worklistc.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

texture <int, 1, hipReadModeElementType> row_offsets;
texture <int, 1, hipReadModeElementType> column_indices;

__global__ void bfs_kernel(int m, DistT *dist, Worklist2 in_queue, Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if(in_queue.pop_id(tid, src)) {
		int row_begin = tex1Dfetch(row_offsets, src);
		int row_end = tex1Dfetch(row_offsets, src + 1);
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = tex1Dfetch(column_indices, offset);
			//DistT new_dist = dist[src] + 1;
			if ((dist[dst] == MYINFINITY) && (atomicCAS(&dist[dst], MYINFINITY, dist[src]+1)==MYINFINITY)) {
			//if (dist[dst] == MYINFINITY) {//Not visited
			//	dist[dst] = new_dist;
				assert(out_queue.push(dst));
			}
		}
	}
}

__global__ void insert(int source, Worklist2 queue) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) queue.push(source);
	return;
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *in_degree, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTexture(0, row_offsets, d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipBindTexture(0, column_indices, d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));

	Worklist2 queue1(nnz), queue2(nnz);
	Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
	int nitems = 1;
	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;

	Timer t;
	t.Start();
	insert<<<1, nthreads>>>(source, *in_frontier);
	nitems = in_frontier->nitems();
	do {
		++ iter;
		nblocks = (nitems - 1) / nthreads + 1;
		bfs_kernel <<<nblocks, nthreads>>> (m, d_dist, *in_frontier, *out_frontier);
		CudaTest("solving failed");
		nitems = out_frontier->nitems();
		Worklist2 *tmp = in_frontier;
		in_frontier = out_frontier;
		out_frontier = tmp;
		out_frontier->reset();
	} while (nitems > 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	return;
}
