#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "topo_vector"
#include "bfs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"
#include <algorithm>

__global__ void initialize(int m, int source, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		expanded[id] = false;
		if(id == source) visited[id] = true;
		else visited[id] = false;
	}
}

__global__ void bfs_kernel(int m, int *row_offsets, int *column_indices, DistT *dist, bool *changed, bool *visited, bool *expanded, int *frontier_size, int depth) {
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int src = warp_id; src < m; src += num_warps) {
		if(visited[src] && !expanded[src]) { // visited but not expanded
			expanded[src] = true;
			// use two threads to fetch Ap[row] and Ap[row+1]
			// this is considerably faster than the straightforward version
			if(thread_lane < 2)
				ptrs[warp_lane][thread_lane] = row_offsets[src + thread_lane];
			const int row_begin = ptrs[warp_lane][0];                   //same as: row_start = row_offsets[row];
			const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = row_offsets[row+1];
			for(int offset = row_begin + thread_lane; offset < row_end; offset += WARP_SIZE) {
				int dst = column_indices[offset];
				//if (dist[dst] > depth) {
				if (dist[dst] == MYINFINITY) {
					//DistT old_dist = atomicMin(&dist[dst], new_dist);
					//if (new_dist < old_dist) {
					dist[dst] = depth;
					*changed = true;
				}
			}
		}
	}
}

__global__ void bfs_update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *in_degree, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	//CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	//CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));
	int *d_frontier_size;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_frontier_size, sizeof(int)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int mblocks = (m - 1) / nthreads + 1;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	const int nSM = deviceProp.multiProcessorCount;
	const int max_blocks_per_SM = maximum_residency(bfs_kernel, nthreads, 0);
	const int max_blocks = max_blocks_per_SM * nSM;
	int nblocks = std::min(max_blocks, DIVIDE_INTO(m, WARPS_PER_BLOCK));
	//int h_frontier_size = 1;
	initialize <<<mblocks, nthreads>>> (m, source, d_visited, d_expanded);
	CudaTest("initializing failed");
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		bfs_kernel <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_dist, d_changed, d_visited, d_expanded, d_frontier_size, iter);
		bfs_update <<<mblocks, nthreads>>> (m, d_dist, d_visited);
		CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//printf("iteration %d: frontier_size = %d\n", iter, h_frontier_size);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	CUDA_SAFE_CALL(hipFree(d_frontier_size));
	return;
}
