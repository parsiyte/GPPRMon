#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "bfs.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#include <thrust/execution_policy.h>
#define BFS_VARIANT "topo_pb"

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByCta(int m, const IndexT *row_offsets, const IndexT *column_indices, const int *front, bool *visited, bool *processed) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int owner;
	__shared__ int sh_src;
	owner = -1;
	int size = 0;
	if(src < m && front[src]) {
		size = row_offsets[src+1] - row_offsets[src];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_src = src;
			processed[src] = 1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_src];
		int row_end = row_offsets[sh_src+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int offset = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[offset];
				visited[dst] = true;
			}
		}
	}
}

__device__ __forceinline__ void expandByWarp(int m, const IndexT *row_offsets, const IndexT *column_indices, const int *front, bool *visited, bool *processed) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_src[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	if(src < m && front[src] && !processed[src]) {
		size = row_offsets[src+1] - row_offsets[src];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_src[warp_id] = src;
			processed[src] = 1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_src[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int offset = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[offset];
				visited[dst] = true;
			}
		}
	}
}

__global__ void push_base(int m, const IndexT *row_offsets, const IndexT *column_indices, const int *front, bool *visited) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && front[src]) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			visited[dst] = true;
		}
	}
}

__global__ void update(int m, DistT *depths, bool *visited, int *front, bool *changed, int depth) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(depths[id] == MYINFINITY && visited[id]) {
			depths[id] = depth;
			front[id] = 1;
			*changed = true;
		}
	}
}

__global__ void push_lb(int m, const IndexT *row_offsets, const IndexT *column_indices, const int *front, bool *visited, bool *processed) {
	expandByCta(m, row_offsets, column_indices, front, visited, processed);
	expandByWarp(m, row_offsets, column_indices, front, visited, processed);
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(src < m && front[src] && !processed[src]) {
		neighbor_offset = row_offsets[src];
		neighbor_size = row_offsets[src+1] - neighbor_offset;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		int offset = gather_offsets[threadIdx.x];
		if(threadIdx.x < total_edges) {
			int dst = column_indices[offset];
			visited[dst] = true;
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *in_degree, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	bool one = 1;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited, *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	int *d_front;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_front, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_front, 0, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(&d_front[source], &one, sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	int iter = 0;
	int nitems = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//nitems = thrust::reduce(thrust::device, d_front, d_front + m, 0, thrust::plus<int>());
		//printf("iteration=%d, num_frontier=%d\n", iter, nitems);
		CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(bool)));
		push_lb <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_front, d_visited, d_processed);
		CUDA_SAFE_CALL(hipMemset(d_front, 0, m * sizeof(int)));
		update <<<nblocks, nthreads>>> (m, d_dist, d_visited, d_front, d_changed, iter);
		CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	CUDA_SAFE_CALL(hipFree(d_front));
	return;
}
