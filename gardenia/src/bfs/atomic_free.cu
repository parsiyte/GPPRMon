#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "atomic_free"
#include "bfs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

__global__ void initialize(int m, int source, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		expanded[id] = false;
		if(id == source) visited[id] = true;
		else visited[id] = false;
	}
}

__global__ void bfs_kernel(int m, int *row_offsets, int *column_indices, DistT *dist, bool *changed, bool *visited, bool *expanded, int *num_frontier, int depth) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		//atomicAdd(num_frontier, 1);
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if (dist[dst] > depth) {
				dist[dst] = depth;
				*changed = true;
			}
		}
	}
}

__global__ void bfs_update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *in_degree, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	//CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	//CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));
	int *d_num_frontier;
	//CUDA_SAFE_CALL(hipMalloc((void **)&d_num_frontier, sizeof(int)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, source, d_visited, d_expanded);
	CudaTest("initializing failed");
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	//int h_num_frontier = 1;

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_num_frontier, &zero, sizeof(int), hipMemcpyHostToDevice));
		bfs_kernel <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_dist, d_changed, d_visited, d_expanded, d_num_frontier, iter);
		bfs_update <<<nblocks, nthreads>>> (m, d_dist, d_visited);
		CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_num_frontier, d_num_frontier, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration %d: num_frontier = %d\n", iter, h_num_frontier);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());

	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	CUDA_SAFE_CALL(hipFree(d_num_frontier));
	return;
}
