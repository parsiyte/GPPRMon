#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#include "bfs.h"
#include "timer.h"
#include "worklistc.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
//#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#define BFS_VARIANT "hybrid_base"

__global__ void bottom_up_kernel(int m, const IndexT *row_offsets, const IndexT *column_indices, const int *front, int *next, DistT *depths, int depth) {
	int dst = blockIdx.x * blockDim.x + threadIdx.x;
	if(dst < m && depths[dst] == MYINFINITY) { // not visited
		int row_begin = row_offsets[dst];
		int row_end = row_offsets[dst+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int src = column_indices[offset];
			if(__ldg(front+src) == 1) { // if the parent is in the current frontier
				depths[dst] = depth;
				next[dst] = 1; // put this vertex into the next frontier
				break;
			}
		}
	}
}

__global__ void top_down_kernel(int m, const IndexT *row_offsets, const IndexT *column_indices, int *degrees, DistT *depths, int *scout_count, Worklist2 in_queue, Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if(in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if ((depths[dst] == MYINFINITY) && (atomicCAS(&depths[dst], MYINFINITY, depths[src]+1)==MYINFINITY)) {
				assert(out_queue.push(dst));
				atomicAdd(scout_count, __ldg(degrees+dst));
			}
		}
	}
}

__global__ void insert(int source, Worklist2 queue) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) queue.push(source);
	return;
}

__global__ void QueueToBitmap(int num, Worklist2 queue, int *bm) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < num) {
		int src;
		if(queue.pop_id(tid, src)) bm[src] = 1;
	}
}

__global__ void BitmapToQueue(int m, int *bm, Worklist2 queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < m && bm[tid]) queue.push(tid);
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *out_row_offsets, int *out_column_indices, int *in_degree, int *h_degree, DistT *h_depths) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_in_row_offsets, *d_in_column_indices;
	int *d_out_row_offsets, *d_out_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_in_row_offsets, in_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_in_column_indices, in_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_row_offsets, out_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_column_indices, out_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	int *d_degree;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_depths;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_depths, h_depths, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_depths[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	int *d_scout_count;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scout_count, sizeof(int)));
	int *front, *next;
	CUDA_SAFE_CALL(hipMalloc((void **)&front, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&next, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(front, 0, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));
	
	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	Worklist2 queue1(m), queue2(m);
	Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
	int alpha = 15, beta = 18;
	int nitems = 1;
	int edges_to_check = nnz;
	int scout_count = h_degree[source];
	insert<<<1, nthreads>>>(source, *in_frontier);

	Timer t;
	t.Start();
	do {
		if(scout_count > edges_to_check / alpha) {
			int awake_count, old_awake_count;
			QueueToBitmap<<<((nitems-1)/512+1), 512>>>(nitems, *in_frontier, front);
			awake_count = nitems;
			do {
				++ iter;
				old_awake_count = awake_count;
				bottom_up_kernel <<<nblocks, nthreads>>> (m, d_in_row_offsets, d_in_column_indices, front, next, d_depths, iter);
				CudaTest("solving bottom_up failed");
				awake_count = thrust::reduce(thrust::device, next, next + m, 0, thrust::plus<int>());
				//printf("BU: (awake_count=%d) ", awake_count);
				//printf("BU: iteration=%d, num_frontier=%d\n", iter, awake_count);
				// swap the queues
				int *temp = front;
				front = next;
				next = temp;
				//thrust::fill(thrust::device, next, next + m, 0);
				CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));
			} while((awake_count >= old_awake_count) || (awake_count > m / beta));
			in_frontier->reset();
			BitmapToQueue<<<((m-1)/512+1), 512>>>(m, front, *in_frontier);
			scout_count = 1;
		} else {
			++ iter;
			edges_to_check -= scout_count;
			nitems = in_frontier->nitems();
			int mblocks = (nitems - 1) / nthreads + 1;
			CUDA_SAFE_CALL(hipMemcpy(d_scout_count, &zero, sizeof(int), hipMemcpyHostToDevice));
			top_down_kernel <<<mblocks, nthreads>>> (m, d_out_row_offsets, d_out_column_indices, d_degree, d_depths, d_scout_count, *in_frontier, *out_frontier);
			CudaTest("solving top_down failed");
			CUDA_SAFE_CALL(hipMemcpy(&scout_count, d_scout_count, sizeof(int), hipMemcpyDeviceToHost));
			nitems = out_frontier->nitems();
			Worklist2 *tmp = in_frontier;
			in_frontier = out_frontier;
			out_frontier = tmp;
			out_frontier->reset();
			//printf("TD: (scout_count=%d) ", scout_count);
			//printf("TD: iteration=%d, num_frontier=%d\n", iter, nitems);
		}
	} while (nitems > 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_depths, d_depths, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_in_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_in_column_indices));
	CUDA_SAFE_CALL(hipFree(d_out_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_out_column_indices));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_scout_count));
	return;
}
