#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#include "bfs.h"
#include "timer.h"
#include "worklistc.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#include <thrust/execution_policy.h>
#define BFS_VARIANT "hybrid_lb"
//#define LB_BU

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void bu_expand_warp(int m, const IndexT *row_offsets, const IndexT *column_indices, DistT *depths, int *front, int *next, int depth) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int dst = id;
	if(dst < m && depths[dst] == MYINFINITY) {
		size = row_offsets[dst+1] - row_offsets[dst];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = dst;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			bool changed = false;
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int src = column_indices[edge];
				if (front[src] == 1) {
					depths[dst] = depth;
					next[dst] = 1;
					changed = true;
				}
			}
			if(__any_sync(0xFFFFFFFF, changed)) break;
		}
	}
}

__global__ void bottom_up_base(int m, const IndexT *row_offsets, const IndexT *column_indices, DistT *depths, int *front, int *next, int depth) {
	//bu_expand_warp(m, row_offsets, column_indices, depths, front, next, depth);
	int dst = blockIdx.x * blockDim.x + threadIdx.x;
	if(dst < m && depths[dst] == MYINFINITY) { // not visited
		int row_begin = row_offsets[dst];
		int row_end = row_offsets[dst+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int src = column_indices[offset];
			if (front[src] == 1) {
				depths[dst] = depth;
				next[dst] = 1;
				break;
			}
		}
	}
}

__global__ void bottom_up_lb(int m, const IndexT *row_offsets, const IndexT *column_indices, DistT *depths, int *front, int *next, int depth) {
	//bu_expand_CTA(m, row_offsets, column_indices, depths, front, next, depth);
	bu_expand_warp(m, row_offsets, column_indices, depths, front, next, depth);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int dst = tid;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	__shared__ int dst_id[BLOCK_SIZE];
	__shared__ bool dstDone[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	dst_id[tx] = 0;
	dstDone[tx] = false;
	
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(dst < m && depths[dst] == MYINFINITY) {
		neighbor_offset = row_offsets[dst];
		neighbor_size = row_offsets[dst+1] - neighbor_offset;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; !dstDone[dst%BLOCK_SIZE] && neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			dst_id[j] = dst;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int dst = dst_id[tx];
			int src = column_indices[edge];
			if (front[src] == 1) {
				depths[dst] = depth;
				next[dst] = 1;
				dstDone[dst%BLOCK_SIZE] = true;
			}
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__device__ void td_expand_CTA(int m, int *row_offsets, int *column_indices, int *degrees, int *scout_count, DistT *depths, Worklist2 &in_queue, Worklist2 &out_queue, int depth) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(in_queue.pop_id(id, vertex)) {
		size = row_offsets[vertex+1] - row_offsets[vertex];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = vertex;
			in_queue.d_queue[id] = -1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			int dst = 0;
			int ncnt = 0;
			if(i < neighbor_size) {
				dst = column_indices[edge];
				if(depths[dst] == MYINFINITY) {
					depths[dst] = depth;
					atomicAdd(scout_count, degrees[dst]);
					ncnt = 1;
				}
			}
			out_queue.push_1item<BlockScan>(ncnt, dst, BLOCK_SIZE);
		}
	}
}

__device__ __forceinline__ void td_expand_warp(int m, int *row_offsets, int *column_indices, int *degrees, int *scout_count, DistT *depths, Worklist2 &in_queue, Worklist2 &out_queue, int depth) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int vertex;
	if(in_queue.pop_id(id, vertex)) {
		if (vertex != -1)
			size = row_offsets[vertex+1] - row_offsets[vertex];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = vertex;
			in_queue.d_queue[id] = -1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int ncnt = 0;
			int dst = 0;
			int edge = row_begin + i;
			if(i < neighbor_size) {
				dst = column_indices[edge];
				if(depths[dst] == MYINFINITY) {
					depths[dst] = depth;
					atomicAdd(scout_count, degrees[dst]);
					ncnt = 1;
				}
			}
			out_queue.push_1item<BlockScan>(ncnt, dst, BLOCK_SIZE);
		}
	}
}

__global__ void top_down_base(int m, int *row_offsets, int *column_indices, int *degrees, int *scout_count, DistT *depths, Worklist2 in_queue, Worklist2 out_queue, int depth) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if(in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if ((depths[dst] == MYINFINITY) && (atomicCAS(&depths[dst], MYINFINITY, depth)==MYINFINITY)) {
				assert(out_queue.push(dst));
				atomicAdd(scout_count, degrees[dst]);
			}
		}
	}
}

__global__ void top_down_lb(int m, int *row_offsets, int *column_indices, int *degrees, int *scout_count, DistT *depths, Worklist2 in_queue, Worklist2 out_queue, int depth) {
	td_expand_CTA(m, row_offsets, column_indices, degrees, scout_count, depths, in_queue, out_queue, depth);
	td_expand_warp(m, row_offsets, column_indices, degrees, scout_count, depths, in_queue, out_queue, depth);
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(in_queue.pop_id(id, vertex)) {
		if(vertex != -1) {
			neighbor_offset = row_offsets[vertex];
			neighbor_size = row_offsets[vertex+1] - neighbor_offset;
		}
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		int ncnt = 0;
		int dst = 0;
		int edge = gather_offsets[threadIdx.x];
		if(threadIdx.x < total_edges) {
			dst = column_indices[edge];
			if(depths[dst] == MYINFINITY) {
				depths[dst] = depth;
				atomicAdd(scout_count, degrees[dst]);
				ncnt = 1;
			}
		}
		out_queue.push_1item<BlockScan>(ncnt, dst, BLOCK_SIZE);
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void insert(int source, Worklist2 queue) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) queue.push(source);
	return;
}

__global__ void QueueToBitmap(int num, Worklist2 queue, int *bm) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num) {
		int src;
		if (queue.pop_id(tid, src)) bm[src] = 1;
	}
}

__global__ void BitmapToQueue(int m, int *bm, Worklist2 queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < m && bm[tid]) queue.push(tid);
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *out_row_offsets, int *out_column_indices, int *in_degree, int *h_degrees, DistT *h_depths) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_in_row_offsets, *d_in_column_indices;
	int *d_out_row_offsets, *d_out_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_in_row_offsets, in_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_in_column_indices, in_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_row_offsets, out_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_column_indices, out_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	int *d_degrees;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degrees, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_degrees, h_degrees, m * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_depths;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_depths, h_depths, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_depths[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	int *d_scout_count;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scout_count, sizeof(int)));
	int *front, *next;
	CUDA_SAFE_CALL(hipMalloc((void **)&front, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&next, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(front, 0, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));
	
	int iter = 0;
	Worklist2 queue1(m), queue2(m);
	Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
	int alpha = 15, beta = 18;
	int nitems = 1;
	int edges_to_check = nnz;
	int scout_count = h_degrees[source];
	
	const int nthreads = BLOCK_SIZE;
	const int nblocks = (m - 1) / nthreads + 1;
	insert<<<1, nthreads>>>(source, *in_frontier);
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		if (scout_count > edges_to_check / alpha) {
			//CUDA_SAFE_CALL(hipMemset(front, 0, m * sizeof(int)));
			int awake_count, old_awake_count;
			QueueToBitmap<<<((nitems-1)/512+1), 512>>>(nitems, *in_frontier, front);
			//awake_count = thrust::reduce(thrust::device, front, front + m, 0, thrust::plus<int>());
			//printf("Transition from TD to BU: nitems=%d, awake_count=%d\n", nitems, awake_count);
			awake_count = nitems;
			do {
				++ iter;
				old_awake_count = awake_count;
#ifdef LB_BU
				bottom_up_lb <<<nblocks, nthreads>>> (m, d_in_row_offsets, d_in_column_indices, d_depths, front, next, iter);
#else
				bottom_up_base <<<nblocks, nthreads>>> (m, d_in_row_offsets, d_in_column_indices, d_depths, front, next, iter);
#endif
				CudaTest("solving bottom_up failed");
				awake_count = thrust::reduce(thrust::device, next, next + m, 0, thrust::plus<int>());
				//printf("BU: (awake_count=%d) ", awake_count);
				//printf("BU: iteration=%d, num_frontier=%d\n", iter, awake_count);
				// swap the queues
				int *temp = front;
				front = next;
				next = temp;
				CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));
			} while((awake_count >= old_awake_count) || (awake_count > m / beta));
			in_frontier->reset();
			BitmapToQueue<<<((m-1)/512+1), 512>>>(m, front, *in_frontier);
			scout_count = 1;
		} else {
			++ iter;
			edges_to_check -= scout_count;
			nitems = in_frontier->nitems();
			const int mblocks = (nitems - 1) / nthreads + 1;
			CUDA_SAFE_CALL(hipMemcpy(d_scout_count, &zero, sizeof(int), hipMemcpyHostToDevice));
			if (1)
				top_down_lb <<<mblocks, nthreads>>> (m, d_out_row_offsets, d_out_column_indices, d_degrees, d_scout_count, d_depths, *in_frontier, *out_frontier, iter);
			else
				top_down_base <<<mblocks, nthreads>>> (m, d_out_row_offsets, d_out_column_indices, d_degrees, d_scout_count, d_depths, *in_frontier, *out_frontier, iter);
			CudaTest("solving top_down failed");
			CUDA_SAFE_CALL(hipMemcpy(&scout_count, d_scout_count, sizeof(int), hipMemcpyDeviceToHost));
			nitems = out_frontier->nitems();
			Worklist2 *tmp = in_frontier;
			in_frontier = out_frontier;
			out_frontier = tmp;
			out_frontier->reset();
			//printf("TD: (scout_count=%d) ", scout_count);
			//printf("TD: iteration=%d, num_frontier=%d\n", iter, nitems);
		}
	} while (nitems > 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_depths, d_depths, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_in_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_in_column_indices));
	CUDA_SAFE_CALL(hipFree(d_out_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_out_column_indices));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_degrees));
	CUDA_SAFE_CALL(hipFree(d_scout_count));
	return;
}
