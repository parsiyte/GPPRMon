#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
#define SYMGS_VARIANT "base"
#include "symgs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

texture<float,1> tex_b;
void bind_b(const float * b) { CUDA_SAFE_CALL(hipBindTexture(NULL, tex_b, b)); }
void unbind_b(const float * b) { CUDA_SAFE_CALL(hipUnbindTexture(tex_b)); }

__global__ void gs_kernel(int num_rows, int * Ap, int * Aj, int* indices, ValueT * Ax, ValueT * x, ValueT * b) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < num_rows) {
		int inew = indices[id];
		int row_begin = Ap[inew];
		int row_end = Ap[inew+1];
		ValueT rsum = 0;
		ValueT diag = 0;
		for (int jj = row_begin; jj < row_end; jj++) {
			const int j = Aj[jj];  //column index
			if (inew == j) diag = Ax[jj];
			else rsum += x[j] * Ax[jj];
		}
		if (diag != 0) x[inew] = (tex1Dfetch(tex_b, inew) - rsum) / diag;
	}
}

void gs_gpu(int *d_Ap, int *d_Aj, int *d_indices, ValueT *d_Ax, ValueT *d_x, ValueT *d_b, int row_start, int row_stop, int row_step) {
	int num_rows = row_stop - row_start;
	const size_t NUM_BLOCKS = (num_rows - 1) / BLOCK_SIZE + 1;
	//printf("num_rows=%d, nblocks=%ld, nthreads=%ld\n", num_rows, NUM_BLOCKS, THREADS_PER_BLOCK);
	gs_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(num_rows, d_Ap, d_Aj, d_indices+row_start, d_Ax, d_x, d_b);
}

void SymGSSolver(int num_rows, int nnz, int *h_Ap, int *h_Aj, int *h_indices, ValueT *h_Ax, ValueT *h_x, ValueT *h_b, std::vector<int> color_offsets) {
	//print_device_info(0);
	Timer t;
	int *d_Ap, *d_Aj, *d_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (num_rows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_indices, sizeof(int) * num_rows));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_indices, h_indices, num_rows * sizeof(int), hipMemcpyHostToDevice));
	ValueT *d_Ax, *d_x, *d_b;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueT) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueT) * num_rows));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_b, sizeof(ValueT) * num_rows));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, num_rows * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, num_rows * sizeof(ValueT), hipMemcpyHostToDevice));
	printf("Launching CUDA SymGS solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	t.Start();
	bind_b(d_b);
	// Forward
	for(size_t i = 0; i < color_offsets.size()-1; i++)
		gs_gpu(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, color_offsets[i], color_offsets[i+1], 1);
	// Backward
	for(size_t i = color_offsets.size()-1; i > 0; i--)
		gs_gpu(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, color_offsets[i-1], color_offsets[i], 1);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	unbind_b(d_b);
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", SYMGS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_x, d_x, sizeof(ValueT) * num_rows, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_indices));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_b));
}

