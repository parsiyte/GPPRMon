#include "hip/hip_runtime.h"
// Copyright 2020
// Author: Xuhao Chen <cxh@mit.edu>
#include <stdio.h>
#include "symgs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

template <int VECTORS_PER_BLOCK, int THREADS_PER_VECTOR>
__global__ void gs_kernel(int m, uint64_t * Ap, int * Aj, 
                          int* indices, ValueT * Ax, 
                          ValueT * x, ValueT * b) {
	__shared__ ValueT sdiags[VECTORS_PER_BLOCK];
	__shared__ ValueT sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2];  // padded to avoid reduction conditionals
	__shared__ IndexT ptrs[VECTORS_PER_BLOCK][2];

	const IndexT THREADS_PER_BLOCK = VECTORS_PER_BLOCK * THREADS_PER_VECTOR;

	const IndexT thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;    // global thread index
	const IndexT thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);          // thread index within the vector
	const IndexT vector_id   = thread_id   /  THREADS_PER_VECTOR;               // global vector index
	const IndexT vector_lane = threadIdx.x /  THREADS_PER_VECTOR;               // vector index within the block
	const IndexT num_vectors = VECTORS_PER_BLOCK * gridDim.x;                   // total number of active vectors

	for(IndexT index = vector_id; index < m; index += num_vectors)
	{
		if(thread_lane == 0) sdiags[vector_lane] = 0; __syncthreads();
		IndexT row = indices[index];

		// use two threads to fetch Ap[row] and Ap[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[vector_lane][thread_lane] = Ap[row + thread_lane];

		const IndexT row_start = ptrs[vector_lane][0];                   //same as: row_start = Ap[row];
		const IndexT row_end   = ptrs[vector_lane][1];                   //same as: row_end   = Ap[row+1];

		// initialize local sum
		ValueT sum = 0;
///*
		if (THREADS_PER_VECTOR == 32 && row_end - row_start > 32) {
			// ensure aligned memory access to Aj and Ax
			IndexT jj = row_start - (row_start & (THREADS_PER_VECTOR - 1)) + thread_lane;

			// accumulate local sums
			if(jj >= row_start && jj < row_end) {
				IndexT col = Aj[jj];
				bool diag = row == col;
				sum += diag ? 0 : Ax[jj] * x[col];
				if(diag) sdiags[vector_lane] = Ax[jj];
			}

			// accumulate local sums
			for(jj += THREADS_PER_VECTOR; jj < row_end; jj += THREADS_PER_VECTOR) {
				IndexT col = Aj[jj];
				bool diag = row == col;
				sum += diag ? 0 : Ax[jj] * x[col];
				if(diag) sdiags[vector_lane] = Ax[jj];
			}
		}
		else {
//*/
			// accumulate local sums
			for(IndexT jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR) {
				IndexT col = Aj[jj];
				bool diag = row == col;
				sum += diag ? 0 : Ax[jj] * x[col];
				if(diag) sdiags[vector_lane] = Ax[jj];
			}
		}

		// store local sum in shared memory
		sdata[threadIdx.x] = sum; __syncthreads();

		// reduce local sums to row sum
		if (THREADS_PER_VECTOR > 16) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads();
		if (THREADS_PER_VECTOR >  8) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		if (THREADS_PER_VECTOR >  4) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		if (THREADS_PER_VECTOR >  2) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		if (THREADS_PER_VECTOR >  1) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();

		// first thread writes the result
		if (thread_lane == 0 && sdiags[vector_lane] != 0)
			x[row] = (b[row] - sdata[threadIdx.x]) / sdiags[vector_lane];
	}
}
size_t nSM;
template <int THREADS_PER_VECTOR>
void gs_gpu(uint64_t *d_Ap, int *d_Aj, 
            int *d_indices, ValueT *d_Ax, 
            ValueT *d_x, ValueT *d_b, 
            int row_start, int row_stop) {
	int m = row_stop - row_start;
	const int VECTORS_PER_BLOCK = BLOCK_SIZE / THREADS_PER_VECTOR;
	//const size_t max_blocks_per_SM = maximum_residency(gs_kernel<VECTORS_PER_BLOCK, THREADS_PER_VECTOR>, BLOCK_SIZE, 0);
	//const size_t max_blocks = max_blocks_per_SM * nSM;
	const int nblocks = std::min(MAX_BLOCKS, DIVIDE_INTO(m, VECTORS_PER_BLOCK));
	//printf("m=%d, nblocks=%d, nthreads=%d, vector_size=%d\n", m, nblocks, BLOCK_SIZE, THREADS_PER_VECTOR);
	gs_kernel<VECTORS_PER_BLOCK, THREADS_PER_VECTOR> <<<nblocks, BLOCK_SIZE>>>(m, d_Ap, d_Aj, d_indices+row_start, d_Ax, d_x, d_b);
}

void gauss_seidel(int m, int nnz, uint64_t *d_Ap, int *d_Aj, 
                  int *d_indices, ValueT *d_Ax, ValueT *d_x, 
                  ValueT *d_b, int row_start, int row_stop, int row_step) {
	int nnz_per_row = nnz / m;
	if (nnz_per_row <=  2) gs_gpu<2>(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, row_start, row_stop);
	else if (nnz_per_row <=  4) gs_gpu<4>(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, row_start, row_stop);
	else if (nnz_per_row <=  8) gs_gpu<8>(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, row_start, row_stop);
	else if (nnz_per_row <= 16) gs_gpu<16>(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, row_start, row_stop);
	else gs_gpu<32>(d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, row_start, row_stop);
}

void SymGSSolver(Graph &g, int *h_indices, 
                 ValueT *h_Ax, ValueT *h_x, 
                 ValueT *h_b, std::vector<int> color_offsets) {
  auto m = g.V();
  auto nnz = g.E();
  auto h_Ap = g.in_rowptr();
  auto h_Aj = g.in_colidx();	
  //print_device_info(0);
  uint64_t *d_Ap;
  VertexId *d_Aj;
	int *d_indices;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (m + 1) * sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_indices, m * sizeof(int)));
  CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_indices, h_indices, m * sizeof(int), hipMemcpyHostToDevice));

	ValueT *d_Ax, *d_x, *d_b;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueT) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_b, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, m* sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_b, h_b, m* sizeof(ValueT), hipMemcpyHostToDevice));
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	nSM = deviceProp.multiProcessorCount;
	printf("Launching CUDA SymGS solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	Timer t;
	t.Start();
	//printf("Forward\n");
	for(size_t i = 0; i < color_offsets.size()-1; i++)
		gauss_seidel(m, nnz, d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, color_offsets[i], color_offsets[i+1], 1);
	//printf("Backward\n");
	for(size_t i = color_offsets.size()-1; i > 0; i--)
		gauss_seidel(m, nnz, d_Ap, d_Aj, d_indices, d_Ax, d_x, d_b, color_offsets[i-1], color_offsets[i], 1);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [cuda_vector] = %f ms.\n", t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_x, d_x, sizeof(ValueT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_indices));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_b));
}

