#include <stdio.h>
#include <stdlib.h>
#include <nvgraph.h>

void check(nvgraphStatus_t status) {
	if (status != NVGRAPH_STATUS_SUCCESS) {
		printf("ERROR : %d\n",status);
		exit(0);
	}
}

int main(int argc, char **argv) {
	const size_t n = 6, nnz = 10, vertex_numsets = 1, edge_numsets = 1;
	float *sssp_1_h;
	void** vertex_dim;
	// nvgraph variables
	nvgraphStatus_t status; nvgraphHandle_t handle;
	nvgraphGraphDescr_t graph;
	nvgraphCSCTopology32I_t CSC_input;
	hipDataType edge_dimT = HIP_R_32F;
	hipDataType* vertex_dimT;
	// Init host data
	sssp_1_h = (float*)malloc(n*sizeof(float));
	vertex_dim = (void**)malloc(vertex_numsets*sizeof(void*));
	vertex_dimT =
		(hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
	CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct
				nvgraphCSCTopology32I_st));
	vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;
	float weights_h[] = {0.333333, 0.5, 0.333333, 0.5, 0.5, 1.0, 0.333333, 0.5, 0.5, 0.5};
	int destination_offsets_h[] = {0, 1, 3, 4, 6, 8, 10};
	int source_indices_h[] = {2, 0, 2, 0, 4, 5, 2, 3, 3, 4};
	check(nvgraphCreate(&handle));
	check(nvgraphCreateGraphDescr (handle, &graph));
	CSC_input->nvertices = n; CSC_input->nedges = nnz;
	CSC_input->destination_offsets = destination_offsets_h;
	CSC_input->source_indices = source_indices_h;
	// Set graph connectivity and properties (tranfers)
	check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input,
				NVGRAPH_CSC_32));
	check(nvgraphAllocateVertexData(handle, graph, vertex_numsets,
				vertex_dimT));
	check(nvgraphAllocateEdgeData (handle, graph, edge_numsets, &edge_dimT));
	check(nvgraphSetEdgeData(handle, graph, (void*)weights_h, 0));
	// Solve
	int source_vert = 0;
	check(nvgraphSssp(handle, graph, 0, &source_vert, 0));
	// Get and print result
	check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));
	//Clean
	free(sssp_1_h); free(vertex_dim);
	free(vertex_dimT); free(CSC_input);
	check(nvgraphDestroyGraphDescr(handle, graph));
	check(nvgraphDestroy(handle));
	return 0;
}
