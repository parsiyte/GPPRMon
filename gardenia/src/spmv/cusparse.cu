// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "spmv.h"
#include "timer.h"
#include "spmv_util.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipblas.h>
#include <hipsparse.h>
#define SPMV_VARIANT "cusparse"

void SpmvSolver(int m, int nnz, IndexT *ApT, IndexT *AjT, ValueT *AxT, IndexT *h_Ap, IndexT *h_Aj, ValueT *h_Ax, ValueT *h_x, ValueT *h_y, int *degrees) { 
	//print_device_info(0);
	int *d_Ap, *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	float *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(float) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(float) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(float) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, m * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, m * sizeof(float), hipMemcpyHostToDevice));
	ValueT *y_copy = (ValueT *)malloc(m * sizeof(ValueT));
	for(int i = 0; i < m; i ++) y_copy[i] = h_y[i];
	SpmvSerial(m, nnz, h_Ap, h_Aj, h_Ax, h_x, y_copy);

	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SpMV solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	const float alpha = 1.0;
	const float beta = 1.0;
	hipsparseMatDescr_t descr = NULL;	
	CudaSparseCheck(hipsparseCreateMatDescr(&descr));
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	hipStream_t streamId;
	hipsparseHandle_t cusparseHandle;
	streamId = NULL;
	cusparseHandle = NULL;
	hipStreamCreateWithFlags(&streamId, hipStreamNonBlocking);
	CudaSparseCheck(hipsparseCreate(&cusparseHandle));
	CudaSparseCheck(hipsparseSetStream(cusparseHandle, streamId));

	Timer t;
	t.Start();
	CudaSparseCheck(hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		m, m, nnz, &alpha, descr, d_Ax, d_Ap, d_Aj, d_x, &beta, d_y));
	CudaTest("solving failed");
	//CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	CudaSparseCheck(hipsparseDestroy(cusparseHandle));
	double time = t.Millisecs();
	float gbyte = bytes_per_spmv(m, nnz);
	float GFLOPs = (time == 0) ? 0 : (2 * nnz / time) / 1e6;
	float GBYTEs = (time == 0) ? 0 : (gbyte / time) / 1e6;
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueT) * m, hipMemcpyDeviceToHost));
	double error = l2_error(m, y_copy, h_y);
	printf("\truntime [%s] = %.4f ms ( %5.2f GFLOP/s %5.1f GB/s) [L2 error %f]\n", SPMV_VARIANT, time, GFLOPs, GBYTEs, error);

	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
}

