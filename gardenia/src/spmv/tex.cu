#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "spmv.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#define SPMV_VARIANT "texture"

// CSR SpMV kernels based on a scalar model (one thread per row)
// spmv_csr_scalar_tex
//   Same as spmv_csr_scalar, except x is accessed via texture cache.
texture<float,1> tex_x;
void bind_x(const float * x) { CUDA_SAFE_CALL(hipBindTexture(NULL, tex_x, x)); }
void unbind_x(const float * x) { CUDA_SAFE_CALL(hipUnbindTexture(tex_x)); }

__global__ void spmv_csr_scalar_kernel(int m, const IndexT * Ap,  const IndexT * Aj, const ValueT * Ax, const ValueT * x, ValueT * y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < m) {
		ValueT sum = y[row];
		int row_begin = Ap[row];
		int row_end = Ap[row+1];
		for (int offset = row_begin; offset < row_end; offset ++) {
			sum += Ax[offset] * tex1Dfetch(tex_x, Aj[offset]);
		}
		y[row] = sum;
	}
}

void SpmvSolver(int m, int nnz, IndexT *ApT, IndexT *AjT, ValueT *AxT, IndexT *h_Ap, IndexT *h_Aj, ValueT *h_Ax, ValueT *h_x, ValueT *h_y, int *degrees) { 
	//print_device_info(0);
	int *d_Ap, *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	ValueT *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueT) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, m * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, m * sizeof(ValueT), hipMemcpyHostToDevice));
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SpMV solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	bind_x(d_x);
	spmv_csr_scalar_kernel <<<nblocks, nthreads>>> (m, d_Ap, d_Aj, d_Ax, d_x, d_y);   
	CudaTest("solving failed");
	unbind_x(d_x);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", SPMV_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
}

