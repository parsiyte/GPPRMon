#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Authors: Xuhao Chen <cxh@mit.edu>
#include "spmv.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void push_base(int m, const uint64_t *Ap,
                          const VertexId *Aj, const ValueT *Ax,
                          const ValueT *x, ValueT *y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < m) {
		int row_begin = Ap[row];
		int row_end = Ap[row+1];
		ValueT value = x[row];
		for (int offset = row_begin; offset < row_end; offset ++){
			IndexT dst = Aj[offset];
			ValueT product = Ax[offset] * value;
			atomicAdd(&y[dst], product);
		}
	}
}

__device__ void __forceinline__ expandByCta(int m, 
                                            const uint64_t *Ap, const VertexId *Aj, 
                                            const ValueT *Ax, const ValueT *x, 
                                            ValueT *y, int *processed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(id < m) {
		size = Ap[id+1] - Ap[id];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = id;
			processed[id] = 1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = Ap[sh_vertex];
		int row_end = Ap[sh_vertex+1];
		int neighbor_size = row_end - row_begin;
		ValueT value = x[sh_vertex];
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = Aj[edge];
				atomicAdd(&y[dst], value * Ax[edge]);
			}
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, 
                                             const uint64_t *Ap, const VertexId *Aj, 
                                             const ValueT *Ax, const ValueT *x, 
                                             ValueT *y, int *processed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	if(id < m && !processed[id]) {
		size = Ap[id+1] - Ap[id];
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = id;
			processed[id] = 1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = Ap[winner];
		int row_end = Ap[winner+1];
		int neighbor_size = row_end - row_begin;
		ScoreT value = x[winner];
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = Aj[edge];
				atomicAdd(&y[dst], value * Ax[edge]);
			}
		}
	}
}

__global__ void push_lb(int m, uint64_t *Ap, VertexId *Aj, 
                        ValueT *Ax, ValueT *x, 
                        ValueT *y, int *processed) {
	expandByCta(m, Ap, Aj, Ax, x, y, processed);
	expandByWarp(m, Ap, Aj, Ax, x, y, processed);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	__shared__ int src_idx[BLOCK_SIZE];
	__shared__ ScoreT values[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	src_idx[tx] = 0;
	values[tx] = 0;
	__syncthreads();

	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if (tid < m && !processed[tid]) {
		neighbor_offset = Ap[tid];
		neighbor_size = Ap[tid+1] - neighbor_offset;
		values[tx] = x[tid];
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	
	int done = 0;
	int neighbors_done = 0;
	while (total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && 
        (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			src_idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int edge = gather_offsets[tx];
			int dst = Aj[edge];
			atomicAdd(&y[dst], values[src_idx[tx]] * Ax[edge]);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

void SpmvSolver(Graph &g, const ValueT* h_Ax, const ValueT *h_x, ValueT *h_y) {
  auto m = g.V();
  auto nnz = g.E();
	auto h_Ap = g.out_rowptr();
	auto h_Aj = g.out_colidx();	
	//print_device_info(0);
	uint64_t *d_Ap;
  VertexId *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (m + 1) * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(VertexId), hipMemcpyHostToDevice));

	ValueT *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueT) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(ValueT) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, m * sizeof(ValueT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, m * sizeof(ValueT), hipMemcpyHostToDevice));

	int *d_processed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_processed, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_processed, 0, m * sizeof(int)));

	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SpMV solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	push_lb <<<nblocks, nthreads>>> (m, d_Ap, d_Aj, d_Ax, d_x, d_y, d_processed);
	//push_base <<<nblocks, nthreads>>> (m, d_Ap, d_Aj, d_Ax, d_x, d_y);
	CudaTest("solving push kernel failed");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [cuda_push] = %f ms.\n", t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
	CUDA_SAFE_CALL(hipFree(d_processed));
}

