#include "hip/hip_runtime.h"
// Copyright (c) 2016, Xuhao Chen
#include "bc.h"
#include "timer.h"
#include "worklistc.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <vector>
#include <hipcub/hipcub.hpp>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#define BC_VARIANT "hybrid_lb"

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
typedef hipcub::BlockReduce<ScoreT, BLOCK_SIZE> BlockReduce;

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__global__ void initialize(int m, int *depths) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) depths[id] = -1;
}

__global__ void insert(Worklist2 in_queue, int src, int *path_counts, int *depths) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id == 0) {
		in_queue.push(src);
		path_counts[src] = 1;
		depths[src] = 0;
	}
	return;
}

__global__ void push_frontier(Worklist2 in_queue, int *queue, int queue_len) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	if (in_queue.pop_id(tid, vertex)) {
		queue[queue_len+tid] = vertex;
	}
}

__global__ void bc_normalize(int m, ScoreT *scores, ScoreT max_score) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < m) scores[tid] = scores[tid] / (max_score);
}

__global__ void set_front(int source, int *front) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id == 0) front[source] = 1;
}

__global__ void BitmapToQueue(int m, int *bm, Worklist2 queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < m && bm[tid]) queue.push(tid);
}

__global__ void QueueToBitmap(int num, Worklist2 queue, int *bm) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num) {
		int src;
		if (queue.pop_id(tid, src)) bm[src] = 1;
	}
}

__device__ __forceinline__ void process_edge(int value, int depth, int dst, const int *degrees, int *depths, int *path_counts, int *scout_count, Worklist2 &out_queue) {
	if ((depths[dst] == -1) && (atomicCAS(&depths[dst], -1, depth) == -1)) {
		assert(out_queue.push(dst));
		atomicAdd(scout_count, __ldg(degrees+dst));
	}
	if (depths[dst] == depth) atomicAdd(&path_counts[dst], value);
}

__global__ void forward_base(const IndexT *row_offsets, const IndexT *column_indices, const int *degrees, int *depths, int *path_counts, int *scout_count, int depth, Worklist2 in_queue, Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if (in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1]; 
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if ((depths[dst] == -1) && (atomicCAS(&depths[dst], -1, depth) == -1)) {
				assert(out_queue.push(dst));
				atomicAdd(scout_count, __ldg(degrees+dst));
			}
			if (depths[dst] == depth) {
				atomicAdd(&path_counts[dst], path_counts[src]);
			}
		}
	}
}

__global__ void forward_push(const IndexT *row_offsets, const IndexT *column_indices, const int *depths, int *path_counts, int *visited, Worklist2 in_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if (in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if (__ldg(depths+dst) == -1) {
				visited[dst] = 1;
				atomicAdd(&path_counts[dst], path_counts[src]);
			}
		}
	}
}

__global__ void forward_lb(const IndexT *row_offsets, const IndexT *column_indices, const int *degrees, int *depths, int *path_counts, int *scout_count, int depth, Worklist2 in_queue, Worklist2 out_queue) {
	//expandByCta(row_offsets, column_indices, path_counts, depths, in_queue, out_queue, depth);
	//expandByWarp(row_offsets, column_indices, path_counts, depths, in_queue, out_queue, depth);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int src;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	__shared__ int srcsrc[SCRATCHSIZE];
	__shared__ int values[BLOCK_SIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(in_queue.pop_id(tid, src)) {
		if(src != -1) {
			neighbor_offset = row_offsets[src];
			neighbor_size = row_offsets[src+1] - neighbor_offset;
			values[tx] = path_counts[src];
		}
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
			srcsrc[scratch_offset + i - done] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int offset = gather_offsets[tx];
			int dst = column_indices[offset];
			process_edge(values[srcsrc[tx]], depth, dst, degrees, depths, path_counts, scout_count, out_queue);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void forward_push_lb(const IndexT *row_offsets, const IndexT *column_indices, const int *depths, int *path_counts, int *visited, Worklist2 in_queue) {
	//expandByCta(row_offsets, column_indices, path_counts, depths, in_queue, out_queue, depth);
	//expandByWarp(row_offsets, column_indices, path_counts, depths, in_queue, out_queue, depth);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int src;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	__shared__ int srcsrc[SCRATCHSIZE];
	__shared__ int values[BLOCK_SIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(in_queue.pop_id(tid, src)) {
		if(src != -1) {
			neighbor_offset = row_offsets[src];
			neighbor_size = row_offsets[src+1] - neighbor_offset;
			values[tx] = path_counts[src];
		}
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
			srcsrc[scratch_offset + i - done] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int offset = gather_offsets[tx];
			int dst = column_indices[offset];
			if (__ldg(depths+dst) == -1) {
				visited[dst] = 1;
				atomicAdd(&path_counts[dst], values[srcsrc[tx]]);
			}
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void forward_pull(int m, const IndexT *row_offsets, const IndexT *column_indices, int *depths, int *path_counts, int depth) {
	int dst = blockIdx.x * blockDim.x + threadIdx.x;
	if(dst < m && depths[dst] == -1) { // not visited
		IndexT row_begin = row_offsets[dst];
		IndexT row_end = row_offsets[dst+1];
		int incoming_total = 0;
		bool is_next = 0;
		for (IndexT offset = row_begin; offset < row_end; ++ offset) {
			IndexT src = column_indices[offset];
			if(depths[src] == depth-1) {
				is_next = 1;
				incoming_total += path_counts[src];
			}
		}
		if (is_next) depths[dst] = depth;
		path_counts[dst] = incoming_total;
	}
}

__global__ void update_pull(const int *depths, int *next, int depth) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(depths[tid] == depth)
		next[tid] = 1;
}

__global__ void update_push(const int *status, const int *degrees, int *depths, Worklist2 queue, int *scout_count, int depth) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(status[tid] == 1) {
	//if(depths[tid] == -1 && status[tid] == 1) {
		depths[tid] = depth;
		queue.push(tid);
		atomicAdd(scout_count, __ldg(degrees+tid));
	}
}

__global__ void reverse_base(int num, const IndexT *row_offsets, const IndexT *column_indices, int start, int *frontiers, ScoreT *scores, const int *path_counts, int *depths, int depth, ScoreT *deltas) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num) {
		int src = frontiers[start + tid];
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if (depths[dst] == depth + 1) {
				deltas[src] += static_cast<ScoreT>(path_counts[src]) / 
					static_cast<ScoreT>(path_counts[dst]) * (1 + deltas[dst]);
			}
		}
		scores[src] += deltas[src];
	}
}

__device__ __forceinline__ void reverse_expand_cta(int num, const IndexT *row_offsets, const IndexT *column_indices, int start, IndexT *frontiers, ScoreT *scores, const int *path_counts, int *depths, int depth, ScoreT *deltas) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	int src = 0;
	int size = 0;
	__shared__ int owner;
	__shared__ int sh_src;
	owner = -1;
	if(tid < num) {
		src = frontiers[start + tid];
		size = row_offsets[src+1] - row_offsets[src];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1) break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_src = src;
			frontiers[start + tid] = -1;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_src];
		int row_end = row_offsets[sh_src+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		int count = path_counts[sh_src];
		ScoreT sum = 0;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int offset = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[offset];
				if(depths[dst] == depth + 1) {
					ScoreT value = static_cast<ScoreT>(count) /
						static_cast<ScoreT>(__ldg(path_counts+dst)) * (1 + deltas[dst]);
					sum += value;
				}
			}
		}
		ScoreT delta_src = BlockReduce(temp_storage).Sum(sum);
		if(threadIdx.x == 0) {
			deltas[sh_src]  = delta_src;
			scores[sh_src] += delta_src;
		}
	}
}

__device__ __forceinline__ void reverse_expand_warp(int num, const IndexT *row_offsets, const IndexT *column_indices, int start, IndexT *frontiers, ScoreT *scores, const int *path_counts, int *depths, int depth, ScoreT *deltas) {
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_src[NUM_WARPS];
	__shared__ ScoreT sdata[BLOCK_SIZE + 16];
	owner[warp_id] = -1;
	int size = 0;
	int src = -1;
	if(tid < num) {
		src = frontiers[start + tid];
		if(src != -1) {
			size = row_offsets[src+1] - row_offsets[src];
		}
	}
	while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_src[warp_id] = src;
			frontiers[start + tid] = -1;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_src[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner+1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		int count = path_counts[winner];
		ScoreT sum = 0;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				int dst = column_indices[edge];
				if(depths[dst] == depth + 1) {
					ScoreT value = static_cast<ScoreT>(count) /
						static_cast<ScoreT>(__ldg(path_counts+dst)) * (1 + deltas[dst]);
					sum += value;
				}
			}
		}
		sdata[threadIdx.x] = sum; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
		if(lane_id == 0) {
			deltas[winner]  = sdata[threadIdx.x];
			scores[winner] += sdata[threadIdx.x];
		}
	}
}

__global__ void reverse_lb(int num, const IndexT *row_offsets, const IndexT *column_indices, int start, IndexT *frontiers, ScoreT *scores, const int *path_counts, int *depths, int depth, ScoreT *deltas) {
	reverse_expand_cta(num, row_offsets, column_indices, start, frontiers, scores, path_counts, depths, depth, deltas);
	reverse_expand_warp(num, row_offsets, column_indices, start, frontiers, scores, path_counts, depths, depth, deltas);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[BLOCK_SIZE];
	//__shared__ int srcs[BLOCK_SIZE];
	__shared__ int idx[BLOCK_SIZE];
	__shared__ int sh_counts[BLOCK_SIZE];
	__shared__ ScoreT sh_deltas[BLOCK_SIZE];
	gather_offsets[tx] = 0;
	//srcs[tx] = 0;
	idx[tx] = 0;
	sh_counts[tx] = 0;
	sh_deltas[tx] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	int src = -1;
	if(tid < num) {
		src = frontiers[start + tid];
		if(src != -1) {
			neighbor_offset = row_offsets[src];
			neighbor_size = row_offsets[src+1] - neighbor_offset;
			sh_counts[tx] = path_counts[src];
		}
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < BLOCK_SIZE; i++) {
			int j = scratch_offset + i - done;
			gather_offsets[j] = neighbor_offset + neighbors_done + i;
			//srcs[j] = src;
			idx[j] = tx;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		if(tx < total_edges) {
			int offset = gather_offsets[tx];
			int dst = column_indices[offset];
			if(depths[dst] == depth + 1) {
				ScoreT value = static_cast<ScoreT>(sh_counts[idx[tx]]) / 
					static_cast<ScoreT>(__ldg(path_counts+dst)) * (1 + deltas[dst]);
				atomicAdd(&sh_deltas[idx[tx]], value); 
			}
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
	__syncthreads();
	if(src != -1) {
		deltas[src]  = sh_deltas[tx];
		scores[src] += sh_deltas[tx];
	}
}

void BCSolver(int m, int nnz, int source, IndexT *in_row_offsets, IndexT *in_column_indices, IndexT *out_row_offsets, IndexT *out_column_indices, int *h_degrees, ScoreT *h_scores) {
	//print_device_info(0);
	int *d_in_row_offsets, *d_in_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_in_row_offsets, in_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_in_column_indices, in_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	int *d_out_row_offsets, *d_out_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_out_row_offsets, out_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_column_indices, out_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores, *d_deltas;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, sizeof(ScoreT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_deltas, sizeof(ScoreT) * m));
	CUDA_SAFE_CALL(hipMemset(d_scores, 0, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemset(d_deltas, 0, m * sizeof(ScoreT)));
	int *d_path_counts, *d_depths, *d_frontiers;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_path_counts, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_frontiers, sizeof(int) * (m+1)));
	CUDA_SAFE_CALL(hipMemset(d_path_counts, 0, m * sizeof(int)));
	//CUDA_SAFE_CALL(hipMemcpy(&d_depths[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	int *d_status;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_status, m * sizeof(int)));
	int *d_degrees;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degrees, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_degrees, h_degrees, m * sizeof(int), hipMemcpyHostToDevice));
	int *d_scout_count;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scout_count, sizeof(int)));
	int *front, *next;
	CUDA_SAFE_CALL(hipMalloc((void **)&front, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&next, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(front, 0, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));

	int depth = 0;
	int nitems = 1;
	int frontiers_len = 0;
	vector<int> depth_index;
	depth_index.push_back(0);
	Worklist2 wl1(m), wl2(m);
	Worklist2 *inwl = &wl1, *outwl = &wl2;
	int nthreads = BLOCK_SIZE;
	int mblocks = (m - 1) / nthreads + 1;
	int alpha = 15, beta = 18;
	int edges_to_check = nnz;
	int scout_count = h_degrees[source];
	//set_front<<<1, 1>>>(source, front);
	initialize<<<mblocks, nthreads>>>(m, d_depths);
	insert<<<1, 1>>>(*inwl, source, d_path_counts, d_depths);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("Launching CUDA BC solver (%d CTAs/SM, %d threads/CTA) ...\n", mblocks, nthreads);

	Timer t;
	t.Start();
	do {
		if(scout_count > edges_to_check / alpha) {
			int awake_count, old_awake_count;
			//QueueToBitmap<<<((nitems-1)/512+1), 512>>>(nitems, *inwl, front);
			awake_count = nitems;
			do {
				++ depth;
				int nblocks = (awake_count - 1) / nthreads + 1;
				push_frontier<<<nblocks, nthreads>>>(*inwl, d_frontiers, frontiers_len);
				frontiers_len += awake_count;
				depth_index.push_back(frontiers_len);
				old_awake_count = awake_count;
				//printf("BU: iteration=%d, num_frontier=%d\n", depth, awake_count);
				forward_pull <<<mblocks, nthreads>>> (m, d_in_row_offsets, d_in_column_indices, d_depths, d_path_counts, depth);
				CudaTest("solving forward failed");
				CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(int)));
				update_pull<<<mblocks, nthreads>>>(d_depths, next, depth);
				awake_count = thrust::reduce(thrust::device, next, next + m, 0, thrust::plus<int>());
				//int *temp = front;
				//front = next;
				//next = temp;
				inwl->reset();
				BitmapToQueue<<<((m-1)/512+1), 512>>>(m, next, *inwl);
			} while((awake_count >= old_awake_count) || (awake_count > m / beta));
			inwl->reset();
			BitmapToQueue<<<((m-1)/512+1), 512>>>(m, next, *inwl);
			scout_count = 1;
			nitems = inwl->nitems();
		} else {
			++ depth;
			nitems = inwl->nitems();
			//printf("TD: iteration=%d, num_frontier=%d\n", depth, nitems);
			CUDA_SAFE_CALL(hipMemset(d_status, 0, m * sizeof(int)));
			edges_to_check -= scout_count;
			int nblocks = (nitems - 1) / nthreads + 1;
			push_frontier<<<nblocks, nthreads>>>(*inwl, d_frontiers, frontiers_len);
			frontiers_len += nitems;
			depth_index.push_back(frontiers_len);
			CUDA_SAFE_CALL(hipMemcpy(d_scout_count, &zero, sizeof(int), hipMemcpyHostToDevice));
			if (nitems <256) {
				forward_lb<<<nblocks, nthreads>>>(d_out_row_offsets, d_out_column_indices, d_degrees, d_depths, d_path_counts, d_scout_count, depth, *inwl, *outwl);
				CudaTest("solving kernel forward failed");
			} else {
				forward_push_lb<<<nblocks, nthreads>>>(d_out_row_offsets, d_out_column_indices, d_depths, d_path_counts, d_status, *inwl);
				CudaTest("solving kernel forward failed");
				update_push<<<mblocks, nthreads>>>(d_status, d_degrees, d_depths, *outwl, d_scout_count, depth);
				CudaTest("solving kernel update failed");
			}
			CUDA_SAFE_CALL(hipMemcpy(&scout_count, d_scout_count, sizeof(int), hipMemcpyDeviceToHost));
			nitems = outwl->nitems();
			Worklist2 *tmp = inwl;
			inwl = outwl;
			outwl = tmp;
			outwl->reset();
		}
	} while (nitems > 0);
	for (int d = depth_index.size() - 2; d >= 0; d--) {
		nitems = depth_index[d+1] - depth_index[d];
		//thrust::sort(thrust::device, d_frontiers+depth_index[d], d_frontiers+depth_index[d+1]);
		int nblocks = (nitems - 1) / nthreads + 1;
		//printf("Reverse: depth=%d, frontier_size=%d\n", d, nitems);
		reverse_lb<<<nblocks, nthreads>>>(nitems, d_out_row_offsets, d_out_column_indices, depth_index[d], d_frontiers, d_scores, d_path_counts, d_depths, d, d_deltas);
		CudaTest("solving kernel reverse failed");
	}
	ScoreT *d_max_score;
	d_max_score = thrust::max_element(thrust::device, d_scores, d_scores + m);
	ScoreT h_max_score;
	CUDA_SAFE_CALL(hipMemcpy(&h_max_score, d_max_score, sizeof(ScoreT), hipMemcpyDeviceToHost));
	nthreads = 512;
	bc_normalize<<<mblocks, nthreads>>>(m, d_scores, h_max_score);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", depth);
	printf("\truntime [%s] = %f ms.\n", BC_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_path_counts));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_deltas));
	CUDA_SAFE_CALL(hipFree(d_frontiers));
	CUDA_SAFE_CALL(hipFree(d_out_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_out_column_indices));
}

