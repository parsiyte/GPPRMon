#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Authors: Xuhao Chen <cxh@mit.edu>
// Topology-driven Minimum Spanning Tree using CUDA
#include "common.h"
#include "timer.h"
#include "graph_io.h"
#include "gbar.h"
#include "component.h"
#include "cuda_launch_config.hpp"
#define MST_TYPE unsigned

__global__ void dinit(int m, MST_TYPE *eleminwts, 
                      MST_TYPE *minwtcomponent, unsigned *partners, 
                      bool *processinnextiteration, 
                      unsigned *goaheadnodeofcomponent) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		eleminwts[id] = MYINFINITY;
		minwtcomponent[id] = MYINFINITY;
		partners[id] = id;
		goaheadnodeofcomponent[id] = m;
		processinnextiteration[id] = false;
	}
}

__global__ void dfindelemin(int m, int *row_offsets, 
                            int *column_indices, WeightT *weight, 
                            MST_TYPE *mstwt, ComponentSpace cs, 
                            MST_TYPE *eleminwts, MST_TYPE *minwtcomponent, 
                            unsigned *partners) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		unsigned src = id;
		unsigned srcboss = cs.find(src);
		unsigned dstboss = m;
		MST_TYPE minwt = MYINFINITY;
		unsigned row_begin = row_offsets[src];
		unsigned row_end = row_offsets[src + 1];
		for (unsigned offset = row_begin; offset < row_end; ++ offset) {
			MST_TYPE wt = (MST_TYPE)weight[offset];
			if (wt < minwt) {
				unsigned dst = column_indices[offset];
				unsigned tempdstboss = cs.find(dst);
				if (srcboss != tempdstboss) {
					minwt = wt;
					dstboss = tempdstboss;
				}
			}
		}
		eleminwts[id] = minwt;
		partners[id] = dstboss;
		if (minwt < minwtcomponent[srcboss] && srcboss != dstboss) {
			atomicMin(&minwtcomponent[srcboss], minwt);
		}
	}
}

__global__ void dfindelemin2(int m, int *row_offsets, 
                             int *column_indices, WeightT *weight, 
                             ComponentSpace cs, MST_TYPE *eleminwts, 
                             MST_TYPE *minwtcomponent, unsigned *partners, 
                             unsigned *goaheadnodeofcomponent) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		unsigned src = id;
		unsigned srcboss = cs.find(src);
		if(eleminwts[id] == minwtcomponent[srcboss] && 
       srcboss != partners[id] && partners[id] != m) {
			unsigned row_begin = row_offsets[src];
			unsigned row_end = row_offsets[src + 1];
			for (unsigned offset = row_begin; offset < row_end; ++ offset) {
				MST_TYPE wt = (MST_TYPE)weight[offset];
				if (wt == eleminwts[id]) {
					unsigned dst = column_indices[offset];
					unsigned tempdstboss = cs.find(dst);
					if (tempdstboss == partners[id]) {
						atomicCAS(&goaheadnodeofcomponent[srcboss], m, id);
					}
				}
			}
		}
	}
}

__global__ void verify_min_elem(int m, int *row_offsets, 
                                int *column_indices, WeightT *weight, 
                                ComponentSpace cs, MST_TYPE *minwtcomponent, 
                                unsigned *partners, bool *processinnextiteration, 
                                unsigned *goaheadnodeofcomponent) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(cs.isBoss(id)) {
			if(goaheadnodeofcomponent[id] == m) {
				return;
			}
			unsigned minwt_node = goaheadnodeofcomponent[id];
			MST_TYPE minwt = minwtcomponent[id];
			if(minwt == MYINFINITY)
				return;
			unsigned row_begin = row_offsets[minwt_node];
			unsigned row_end = row_offsets[minwt_node + 1];
			for (unsigned offset = row_begin; offset < row_end; ++ offset) {
				MST_TYPE wt = (MST_TYPE)weight[offset];
				if (wt == minwt) {
					unsigned dst = column_indices[offset];
					unsigned tempdstboss = cs.find(dst);
					if(tempdstboss == partners[minwt_node] && tempdstboss != id) {
						processinnextiteration[minwt_node] = true;
						return;
					}
				}
			}
		}
	}
}

__device__ volatile int g_mutex;
__device__ void __gpu_sync_atomic(int goalVal) {
	int tid = threadIdx.x * blockDim.y + threadIdx.y;
	__threadfence();
	__syncthreads();
	if (tid == 0) {
		atomicAdd((int *)&g_mutex, 1);
		while(g_mutex % goalVal != 0) {} // Note: this causes GPGPU-Sim not terminating, need to implement 'volatile' in GPGPU-Sim
	}
	__syncthreads();
}

__global__ void dfindcompmintwo(int m, unsigned *mstwt, ComponentSpace csw, 
                                MST_TYPE *eleminwts, MST_TYPE *minwtcomponent, 
                                unsigned *partners, bool *processinnextiteration, 
                                GlobalBarrier gb, bool *repeat, unsigned *count) {
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id, nthreads = blockDim.x * gridDim.x;
	unsigned up = (m + nthreads - 1) / nthreads * nthreads;
	unsigned srcboss, dstboss;
	for(id = tid; id < up; id += nthreads) {
		if(id < m && processinnextiteration[id]) {
			srcboss = csw.find(id);
			dstboss = csw.find(partners[id]);
		}
		//gb.Sync();
		__gpu_sync_atomic(gridDim.x);
		if (id < m && processinnextiteration[id] && srcboss != dstboss) {
			//printf("trying unify id=%d (%d -> %d)\n", id, srcboss, dstboss);
			if (csw.unify(srcboss, dstboss)) {
				atomicAdd(mstwt, eleminwts[id]);
				atomicAdd(count, 1);
				//printf("u %d -> %d (%d)\n", srcboss, dstboss, eleminwts[id]);
				processinnextiteration[id] = false;
				eleminwts[id] = MYINFINITY;	// mark end of processing to avoid getting repeated.
			}
			else {
				*repeat = true;
			}
			//printf("\tcomp[%d] = %d.\n", srcboss, csw.find(srcboss));
		}
		//gb.Sync();
		__gpu_sync_atomic(gridDim.x);
	}
}

int main(int argc, char *argv[]) {
	printf("Minimum Spanning Tree by Xuhao Chen\n");
	if (argc < 2) {
		printf("Usage: %s <graph>\n", argv[0]);
		exit(1);
	}
	int m, n, nnz, *h_row_offsets = NULL, *h_column_indices = NULL, *h_degree = NULL;
	WeightT *h_weight = NULL;
	read_graph(argc, argv, m, n, nnz, h_row_offsets, h_column_indices, h_degree, h_weight, true);
	print_device_info(0);
	WeightT *d_weight;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_weight, nnz * sizeof(WeightT)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_weight, h_weight, nnz * sizeof(WeightT), hipMemcpyHostToDevice));
	int mutex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_mutex), &mutex, sizeof(int)));
	
	MST_TYPE *mstwt, hmstwt = 0;
	int iteration = 0;
	unsigned *partners;
	MST_TYPE *eleminwts, *minwtcomponent;
	bool *processinnextiteration;
	unsigned *goaheadnodeofcomponent;
	ComponentSpace cs(m);
	unsigned prevncomponents, currncomponents = m;
	bool repeat = false, *grepeat;
	unsigned edgecount = 0, *gedgecount;

	CUDA_SAFE_CALL(hipMalloc((void **)&mstwt, sizeof(MST_TYPE)));
	CUDA_SAFE_CALL(hipMemcpy(mstwt, &hmstwt, sizeof(MST_TYPE), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc((void **)&eleminwts, m * sizeof(MST_TYPE)));
	CUDA_SAFE_CALL(hipMalloc((void **)&minwtcomponent, m * sizeof(MST_TYPE)));
	CUDA_SAFE_CALL(hipMalloc((void **)&partners, m * sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc((void **)&processinnextiteration, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&goaheadnodeofcomponent, m * sizeof(unsigned)));
	CUDA_SAFE_CALL(hipMalloc(&grepeat, sizeof(bool) * 1));
	CUDA_SAFE_CALL(hipMemcpy(grepeat, &repeat, sizeof(bool) * 1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc(&gedgecount, sizeof(unsigned) * 1));
	CUDA_SAFE_CALL(hipMemcpy(gedgecount, &edgecount, sizeof(unsigned) * 1, hipMemcpyHostToDevice));

	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	int nSM = 13;
	//const size_t max_blocks = maximum_residency(dfindcompmintwo, nthreads, 0);
	int max_blocks = 1;
	printf("Setup global barrier, max_blocks=%d\n", max_blocks);
	GlobalBarrierLifetime gb;
	gb.Setup(nSM * max_blocks);
	printf("Finding mst...\n");
	Timer t;
	t.Start();
	do {
		++iteration;
		prevncomponents = currncomponents;
		dinit<<<nblocks, nthreads>>>(m, eleminwts, minwtcomponent, partners, processinnextiteration, goaheadnodeofcomponent);
		CudaTest("dinit failed");
		dfindelemin<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_weight, mstwt, cs, eleminwts, minwtcomponent, partners);
		dfindelemin2<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_weight, cs, eleminwts, minwtcomponent, partners, goaheadnodeofcomponent);
		verify_min_elem<<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_weight, cs, minwtcomponent, partners, processinnextiteration, goaheadnodeofcomponent);
		CudaTest("dfindelemin failed");
		do {
			repeat = false;
			CUDA_SAFE_CALL(hipMemcpy(grepeat, &repeat, sizeof(bool) * 1, hipMemcpyHostToDevice));
			dfindcompmintwo <<<nSM * max_blocks, nthreads>>> (m, mstwt, cs, eleminwts, minwtcomponent, partners, processinnextiteration, gb, grepeat, gedgecount);
			CudaTest("dfindcompmintwo failed");
			CUDA_SAFE_CALL(hipMemcpy(&repeat, grepeat, sizeof(bool) * 1, hipMemcpyDeviceToHost));
		} while (repeat); // only required for quicker convergence?
		currncomponents = cs.numberOfComponentsHost();
		CUDA_SAFE_CALL(hipMemcpy(&hmstwt, mstwt, sizeof(hmstwt), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(&edgecount, gedgecount, sizeof(unsigned) * 1, hipMemcpyDeviceToHost));
		printf("\titeration %d, number of components = %d (%d), mstwt = %u mstedges = %u\n", iteration, currncomponents, prevncomponents, hmstwt, edgecount);
	} while (currncomponents != prevncomponents);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\tmstwt = %u, iterations = %d.\n", hmstwt, iteration);
	printf("\t%s result: weight: %u, components: %u, edges: %u\n", argv[1], hmstwt, currncomponents, edgecount);
	printf("\truntime [mst] = %f ms.\n", t.Millisecs());
	return 0;
}
