#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "cc.h"
#include "gbar.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>
#define CC_VARIANT "fusion"

__device__ void hook(int m, int *row_offsets, int *column_indices, CompT *comp, bool *changed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if(src < m) {
			int comp_src = comp[src];
			int row_begin = row_offsets[src];
			int row_end = row_offsets[src+1];
			for (int offset = row_begin; offset < row_end; ++ offset) {
				int dst = column_indices[offset];
				//int comp_dst = comp[dst];
				int comp_dst = __ldg(comp+dst);
				if (comp_src == comp_dst) continue;
				int high_comp = comp_src > comp_dst ? comp_src : comp_dst;
				int low_comp = comp_src + (comp_dst - high_comp);
				if (high_comp == comp[high_comp]) {
					*changed = true;
					comp[high_comp] = low_comp;
				}
			}
		}
	}
}

__device__ void shortcut(int m, int *row_offsets, int *column_indices, CompT *comp) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int total_inputs = (m - 1) / (gridDim.x * blockDim.x) + 1;
	for (int src = tid; total_inputs > 0; src += blockDim.x * gridDim.x, total_inputs--) {
		if(src < m) {
			while (comp[src] != comp[comp[src]]) {
				comp[src] = comp[comp[src]];
			}
		}
	}
}

__global__ void cc_kernel(int m, int *row_offsets, int *column_indices, CompT *comp, bool *changed, GlobalBarrier gb) {
	while (*changed) {
		*changed = false;
		hook(m, row_offsets, column_indices, comp, changed);
		gb.Sync();
		shortcut(m, row_offsets, column_indices, comp);
		gb.Sync();
	}
}

void CCSolver(int m, int nnz, IndexT *in_row_offsets, IndexT *in_column_indices, IndexT *h_row_offsets, int *h_column_indices, int *degrees, CompT *h_comp, bool is_directed) {
	//print_device_info(0);
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CompT *d_comp;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_comp, sizeof(CompT) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_comp, h_comp, m * sizeof(CompT), hipMemcpyHostToDevice));
	bool h_changed, *d_changed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));

	int iter = 0;
	const int nthreads = BLOCK_SIZE;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	const int nSM = deviceProp.multiProcessorCount;
	const int max_blocks_per_SM = maximum_residency(cc_kernel, nthreads, 0);
	const int max_blocks = max_blocks_per_SM * nSM;
	const int nblocks = std::min(max_blocks, DIVIDE_INTO(m, nthreads));
	GlobalBarrierLifetime gb;
	gb.Setup(nblocks);
	printf("Launching CUDA CC solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	h_changed = true;
	CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(h_changed), hipMemcpyHostToDevice));
	cc_kernel<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_comp, d_changed, gb);
	CudaTest("solving cc_kernel failed");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", CC_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_comp, d_comp, sizeof(CompT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_changed));
}

