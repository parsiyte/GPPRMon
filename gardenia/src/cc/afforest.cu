#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include "cc.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"

__device__ void link(VertexId u, VertexId v, IndexT *comp) {
	IndexT p1 = comp[u];
	IndexT p2 = comp[v];
	while (p1 != p2) {
		IndexT high = p1 > p2 ? p1 : p2;
		IndexT low = p1 + (p2 - high);
		IndexT p_high = comp[high];
		if ((p_high == low) || (p_high == high && 
        atomicCAS(&comp[high], high, low) == high))
			break;
		p1 = comp[comp[high]];
		p2 = comp[low];
	}
}

__global__ void compress(int m, CompT *comp) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		while (comp[src] != comp[comp[src]]) {
			comp[src] = comp[comp[src]];
		}
	}
}

__global__ void afforest(int m, const uint64_t* row_offsets, 
                         const VertexId* column_indices, 
                         CompT *comp, int32_t r) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1]; 
		int start_offset = min(r, row_end - row_begin);
		row_begin += start_offset;
		for (IndexT offset = row_begin; offset < row_end; offset ++) {
			IndexT dst = column_indices[offset];
			link(src, dst, comp);
			break;
		}
	}
}

__global__ void afforest_undirected(int m, int c, 
                                    const uint64_t* row_offsets, 
                                    const VertexId* column_indices, 
                                    CompT *comp, int r) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && comp[src] != c) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1]; 
		int start_offset = min(r, row_end - row_begin);
		row_begin += start_offset;
		for (IndexT offset = row_begin; offset < row_end; offset ++) {
			IndexT dst = column_indices[offset];
			link(src, dst, comp);
		}
	}
}

__global__ void afforest_directed(int m, int c, 
                                  const uint64_t* in_row_offsets, 
                                  const VertexId* in_column_indices, 
                                  const uint64_t* row_offsets, 
                                  const VertexId* column_indices, 
                                  CompT *comp, int r) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m && comp[src] != c) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1]; 
		int start_offset = min(r, row_end - row_begin);
		row_begin += start_offset;
		for (IndexT offset = row_begin; offset < row_end; offset ++) {
			IndexT dst = column_indices[offset];
			link(src, dst, comp);
		}
		row_begin = in_row_offsets[src];
		row_end = in_row_offsets[src+1];
		for (IndexT offset = row_begin; offset < row_end; offset ++) {
			IndexT dst = in_column_indices[offset];
			link(src, dst, comp);
		}
	}
}

void CCSolver(Graph &g, CompT *h_comp) {
  auto m = g.V();
  auto nnz = g.E();
	auto in_row_offsets = g.in_rowptr();
	auto out_row_offsets = g.out_rowptr();
	auto in_column_indices = g.in_colidx();	
	auto out_column_indices = g.out_colidx();	
	//print_device_info(0);
	uint64_t *d_in_row_offsets, *d_out_row_offsets;
	VertexId *d_in_column_indices, *d_out_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_row_offsets, (m + 1) * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_in_column_indices, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_row_offsets, (m + 1) * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_out_column_indices, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMemcpy(d_in_row_offsets, in_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_in_column_indices, in_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_row_offsets, out_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_out_column_indices, out_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
	CompT *d_comp;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_comp, m * sizeof(CompT)));
	CUDA_SAFE_CALL(hipMemcpy(d_comp, h_comp, m * sizeof(CompT), hipMemcpyHostToDevice));

	int neighbor_rounds = 2;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA CC solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	for (int r = 0; r < neighbor_rounds; ++r) {
		afforest<<<nblocks, nthreads>>>(m, d_out_row_offsets, d_out_column_indices, d_comp, r);
		CudaTest("solving kernel afforest failed");
		compress<<<nblocks, nthreads>>>(m, d_comp);
		CudaTest("solving kernel compress failed");
	}
	CUDA_SAFE_CALL(hipMemcpy(h_comp, d_comp, m * sizeof(CompT), hipMemcpyDeviceToHost));
	IndexT c = SampleFrequentElement(m, h_comp);
	if (!g.is_directed()) {
		afforest_undirected<<<nblocks, nthreads>>>(m, c, d_out_row_offsets, d_out_column_indices, d_comp, neighbor_rounds);
	} else {
		afforest_directed<<<nblocks, nthreads>>>(m, c, d_in_row_offsets, d_in_column_indices, d_out_row_offsets, d_out_column_indices, d_comp, neighbor_rounds);
	}
	compress<<<nblocks, nthreads>>>(m, d_comp);
	CudaTest("solving kernel compress failed");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [cuda_afforest] = %f ms.\n", t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_comp, d_comp, m * sizeof(CompT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_in_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_out_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_in_column_indices));
	CUDA_SAFE_CALL(hipFree(d_out_column_indices));
}

